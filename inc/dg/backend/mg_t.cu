#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>
#include "xspacelib.cuh"
#include "interpolation.cuh"
#include "../blas.h"
#include "evaluation.cuh"

#include <math.h>

double function( double x, double y)
{   return sin(x)*cos(y/4.);
}
typedef cusp::coo_matrix<int, double, cusp::host_memory> Matrix;

int main()
{   double x0 = 0., x1 = 2.*M_PI;
    int p = 3, n = 4;

    dg::Grid2d grid_2h(x0, x1, x0, x1, p, n/2, n/2);
    dg::DVec vec_2h = dg::evaluate( function, grid_2h);
    for( unsigned i=0; i<vec_2h.size(); ++i)
    {   std::cout<< vec_2h[i] << std::endl;
    }
    std::cout << "----------" <<std::endl;
    dg::Grid2d grid_1h(x0, x1, x0, x1, p, n, n);
    Matrix C2F = dg::create::interpolation( grid_1h, grid_2h);
    dg::DVec vec_1h = dg::evaluate( dg::one, grid_1h);
    dg::blas2::symv( C2F, vec_2h, vec_1h);
//    for( unsigned i=0; i<vec_1h.size(); ++i)
//    {   std::cout<< vec_1h[i] << std::endl;
//    }
//    std::cout << "----------" <<std::endl;
    Matrix F2C = dg::create::interpolation( grid_2h, grid_1h);
    dg::blas2::symv( F2C, vec_1h, vec_2h);
    for( unsigned i=0; i<vec_2h.size(); ++i)
    {   std::cout<< vec_2h[i] << std::endl;
    }


//    dg::Grid1d grid_2h(x0, x1, p, n/2);
//    dg::DVec vec_2h = dg::evaluate( function, grid_2h);
//    for( unsigned i=0; i<vec_2h.size(); ++i)
//      {   std::cout<< vec_2h[i] << std::endl;
//      }
//    std::cout << "----------" <<std::endl;
//    dg::Grid1d grid_1h(x0, x1, p, n);
//    Matrix C2F = dg::create::interpolation( grid_1h, grid_2h);
//    dg::DVec vec_1h = dg::evaluate( dg::one, grid_1h);
//    dg::blas2::symv( C2F, vec_2h, vec_1h);
//    for( unsigned i=0; i<vec_1h.size(); ++i)
//    {   std::cout<< vec_1h[i] << std::endl;
//    }
//    std::cout << "----------" <<std::endl;
//    Matrix F2C = dg::create::interpolation( grid_2h, grid_1h);
//    dg::blas2::symv( F2C, vec_1h, vec_2h);
//    for( unsigned i=0; i<vec_2h.size(); ++i)
//      {   std::cout<< vec_2h[i] << std::endl;
//      }
    return 0;
}
