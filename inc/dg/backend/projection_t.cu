#include "hip/hip_runtime.h"
#include <iostream>
#include <cusp/print.h>
#include "projection.cuh"
#include "evaluation.cuh"
#include "blas.h"
#include "typedefs.cuh"

double sine( double x){ return sin(x);}
double sine( double x, double y){return sin(x)*sin(y);}

int main()
{
    std::cout << "TEST 1D\n";
    unsigned n_old = 4, n_new = 3, N_old = 10, N_new = 1;
    std::cout << "Type n and N of old (fine) grid!\n";
    std::cin >> n_old >> N_old;
    std::cout << "Type n and N of new (coarser) grid!\n";
    std::cin >> n_new >> N_new;
    dg::Grid1d go ( 0, M_PI, n_old, N_old);
    dg::Grid1d gn ( 0, M_PI, n_new, N_new);
    cusp::coo_matrix<int, double, cusp::host_memory> proj = dg::create::transformation( gn, go);
    cusp::coo_matrix<int, double, cusp::host_memory> inte = dg::create::interpolation( gn, go);
    thrust::host_vector<double> v = dg::evaluate( sine, go);
    thrust::host_vector<double> w1do = dg::create::weights( go);
    thrust::host_vector<double> w1dn = dg::create::weights( gn);
    dg::HVec oneo( go.size(), 1.);
    dg::HVec onen( gn.size(), 1.);
    thrust::host_vector<double> w( gn.size());
    dg::blas2::gemv( proj, v, w);
    std::cout << "Original vector  "<<dg::blas2::dot( oneo, w1do, v) << "\n";
    std::cout << "Projected vector "<<dg::blas2::dot( onen, w1dn, w) << "\n";
    std::cout << "Difference       "<<dg::blas2::dot( oneo, w1do, v) - dg::blas2::dot( onen, w1dn, w) << "\n"<<std::endl;
    dg::blas2::gemv( inte, v, w);
    std::cout << "Original vector  "<<dg::blas2::dot( oneo, w1do, v) << "\n";
    std::cout << "Interpolated vec "<<dg::blas2::dot( onen, w1dn, w) << "\n";
    std::cout << "Difference       "<<dg::blas2::dot( oneo, w1do, v) - dg::blas2::dot( onen, w1dn, w) << "\n"<<std::endl;

    std::cout << "TEST 2D\n";
    
    dg::Grid2d g2o (0, M_PI, 0, M_PI, n_old, N_old, N_old);
    dg::Grid2d g2n (0, M_PI, 0, M_PI, n_new, N_new, N_new);
    cusp::coo_matrix<int, double, cusp::host_memory> proj2d = dg::create::transformation( g2n, g2o);
    cusp::coo_matrix<int, double, cusp::host_memory> inte2d = dg::create::interpolation( g2n, g2o);
    const dg::HVec sinO = dg::evaluate( sine, g2o), 
                   sinN = dg::evaluate( sine, g2n);
    dg::HVec w2do = dg::create::weights( g2o);
    dg::HVec w2dn = dg::create::weights( g2n);
    dg::HVec sinP( g2n.size());
    dg::blas2::gemv( proj2d, sinO, sinP);
    std::cout << "Original vector     "<<sqrt(dg::blas2::dot( sinO, w2do, sinO)) << "\n";
    std::cout << "Projected vector    "<<sqrt(dg::blas2::dot( sinP, w2dn, sinP)) << "\n";
    std::cout << "Difference in Norms "<<sqrt(dg::blas2::dot( sinO, w2do, sinO)) - sqrt(dg::blas2::dot( sinP, w2dn, sinP)) << std::endl;
    std::cout << "Difference between projection and evaluation      ";
    dg::blas1::axpby( 1., sinN, -1., sinP);
    std::cout << sqrt(dg::blas2::dot( sinP, w2dn, sinP)/dg::blas2::dot(sinN, w2dn, sinN))<<"\n";
    dg::blas2::gemv( inte2d, sinO, sinP);
    std::cout << "Interpolated vec    "<<sqrt(dg::blas2::dot( sinP, w2dn, sinP)) << "\n";
    std::cout << "Difference in Norms "<<sqrt(dg::blas2::dot( sinO, w2do, sinO)) - sqrt(dg::blas2::dot( sinP, w2dn, sinP)) << "\n" << std::endl;

    return 0;
}
