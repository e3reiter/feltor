#include "hip/hip_runtime.h"
#include <iostream>
#include "chebyshev.h"
#include "elliptic.h"
#include "backend/timer.cuh"

const double lx = M_PI;
const double ly = 2.*M_PI;

double fct(double x, double y)
{   return sin(y)*sin(x+M_PI/2.);
}
double derivative( double x, double y)
{   return cos(x+M_PI/2.)*sin(y);
}
double laplace_fct( double x, double y)
{   return 2*sin(y)*sin(x+M_PI/2.);
}
dg::bc bcx = dg::NEU;
double initial( double x, double y)
{   return sin(0);
}

int main()
{   dg::Timer t;
    unsigned n, Nx, Ny;
    std::cout << "Type n, Nx, Ny";
    std::cin >> n >> Nx >> Ny;
    dg::Grid2d grid( 0., lx, 0, ly, n, Nx, Ny, bcx, dg::PER);
    std::cout<<"Evaluate initial condition...\n";
    dg::DVec x = dg::evaluate( initial, grid);

    std::cout << "Create Laplacian...\n";
    t.tic();
    dg::DMatrix DX = dg::create::dx( grid);
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> lap( grid, dg::not_normed, dg::forward);
    t.toc();
    std::cout<< "Creation took "<<t.diff()<<"s\n";

    return 0;
}
