#include "hip/hip_runtime.h"
#include <iostream>
#include "manualmatrix.h"
#include "analysematrix.h"
#include "elliptic.h"
#include "backend/timer.cuh"

/* test analysematrix.h for available classes */

const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);    //must be strictly positive
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}

int main()
{   dg::Timer t;
    // for some random symmetric positive matrix
    int n_rand, div;
//    std::cout<< "n for random psd and divisor, please:" <<std::endl;
//    std::cin>> n_rand >> div;
//    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
//    std::cout<< "constructing random psd ..." <<std::endl;
//    dg::RandPSDmatrix<dg::DVec> spd(n_rand, 1.1);
//    dg::AnalysisMatrix<dg::DVec> am(n_rand, n_rand);
    double ev_max;
//    am( spd, div, ev_max);
    // now for the elliptic object
    unsigned p, Nx, Ny;
    double eps, jfactor;
    std::cout << "Type p, Nx and Ny and epsilon and jfactor (1), new divisor, too! \n";
    std::cin >> p >> Nx >> Ny;
    std::cin >> eps >> jfactor >> div;
    dg::Grid2d grid( 0, lx, 0, ly, p, Nx, Ny, bcx, bcy);
    dg::DVec chi =  dg::evaluate( pol, grid);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout << "Create Polarisation object and set chi!\n";
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> ell( grid, dg::not_normed, dg::centered, jfactor);
    ell.set_chi( chi);
    dg::AnalysisMatrix<dg::DVec> ae(p*p*Nx*Ny, p*p*Nx*Ny);
    ae( ell, div, ev_max);
    return 0;
}
