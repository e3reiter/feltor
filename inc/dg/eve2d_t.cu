#include "hip/hip_runtime.h"
/* check EVEs capability to estimate the maximum Eigenvalue of Matrices */

#include <iostream>
#include "manualmatrix.h"
#include "eigenmatrix.h"
#include "eve.h"
#include "elliptic.h"
#include "blas.h"

// Imitate "difficult" physical problem
const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}


int main()
{   // random SPD matrix:
    //   generate Matrix
    int n, div;
    double seed;
    std::cout << "Type n, seed and divisor:" <<std::endl;
    std::cin >> n >> seed >> div;
    dg::RandPSDmatrix<dg::DVec> A(n, seed);
    //   solve with decomposition
    dg::EVarbitraryMatrix<dg::DVec> A_decomp(n, div);
    double ev_decomp;
    A_decomp( A, ev_decomp);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via decomposition: " << ev_decomp <<std::endl;
    //   solve with EVE
    //   ... needs dummy x and b as EVE works on solving Ax = b
    dg::DVec x(n, 0.0), b(n, 1.0);
    dg::EVE<dg::DVec> A_eve( x, n);
    double ev_eve;
    unsigned niter = A_eve( A, x, b, ev_eve);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via EVEs guess: " << ev_eve << " after "<< niter << " iterations"<<std::endl;
    std::cout<< "+ + + + + + + + + + + + + + +" <<std::endl;
    std::cout<< "decomp-EVE: " << ev_decomp - ev_eve <<std::endl;
    std::cout<< "+ + + + + + + + + + + + + + +" <<std::endl;

    // for an elliptic object
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    unsigned Nx, Ny, p;
    double jfactor;
    std::cout << "Type p, Nx and Ny and jfactor (1) as well as divisor! \n";
    std::cin >> p >> Nx >> Ny; //more N means less iterations for same error
    std::cin >> jfactor >> div;
    dg::Grid2d grid( 0, lx, 0, ly, p, Nx, Ny, bcx, bcy);
    dg::DVec w2d = dg::create::weights( grid);
    dg::DVec v2d = dg::create::inv_weights( grid);
    dg::DVec one = dg::evaluate( dg::one, grid);
    b = dg::evaluate( rhs, grid);
    dg::DVec chi =  dg::evaluate( pol, grid);
    dg::DVec chi_inv(chi);
    dg::blas1::transform( chi, chi_inv, dg::INVERT<double>());
    dg::blas1::pointwiseDot( chi_inv, v2d, chi_inv);
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> ell( grid, dg::not_normed, dg::centered, jfactor);
    ell.set_chi( chi);
    std::cout << "Created Polarisation object and set chi!\n";
    //   solve with decomposition
    dg::EVarbitraryMatrix<dg::DVec> ell_decomp(p*p*Nx*Ny, div);
    ell_decomp( ell, ev_decomp);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via decomposition: " << ev_decomp <<std::endl;
    //   solve with EVE
    x = dg::evaluate( initial, grid);
    dg::EVE<dg::DVec> ell_eve( x, p*p*Nx*Ny);
    double ev_max;
    niter = ell_eve( ell, x, b, ev_max);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via EVEs guess: " << ev_max << " after "<< niter << " iterations"<<std::endl;
    //    does it work with eInvert (tested as mere wrapper for EVE)
    b = dg::evaluate( rhs, grid);
    x = dg::evaluate( initial, grid);
    dg::eInvert<dg::DVec> invert( x, p*p*Nx*Ny, 1e-6, 1, false);
    niter = invert( ell, x, b, w2d, chi_inv, v2d, ev_eve);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via eInvert guess: " << ev_eve << " after "<< niter << " iterations"<<std::endl;
    return 0;
}
