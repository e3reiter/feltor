#include "hip/hip_runtime.h"
/* check EVEs capability to estimate the maximum Eigenvalue of Matrices */

#include <iostream>
#include "manualmatrix.h"
#include "eigenmatrix.h"
#include "eve.h"
#include "elliptic.h"

// Imitate "difficult" physical problem
const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}

template< class Vector>
void printvector( Vector& v)
{ for( uint i=0; i<v.size(); ++i)
    {   std::cout<<v[i]<<std::endl;
    }
  std::cout<<"- - - - - - - -"<<std::endl;
}

int main()
{   // random SPD matrix:
    //   generate Matrix
    int n;
    double seed;
    std::cout << "Type n and seed:" <<std::endl;
    std::cin >> n >> seed;
    dg::RandPSDmatrix<dg::DVec> A(n, seed);
    //   solve with decomposition
    dg::EVarbitraryMatrix A_decomp(n);
    dg::DVec evt_decomp(1, 0.0), evb_decomp(0, 0.0);;
    A_decomp( A, evt_decomp, evb_decomp);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via decomposition: " <<std::endl;
    printvector<dg::DVec> (evt_decomp);
    //   solve with EVE
    //   ... needs dummy x and b as EVE works on solving Ax = b
    dg::DVec x(n, 0.0), b(n, 1.0);
    dg::EVE<dg::DVec> A_eve( x, n);
    double ev_eve;
    unsigned niter = A_eve( A, x, b, ev_eve);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via EVEs guess: " << ev_eve << " after "<< niter << " iterations"<<std::endl;
    std::cout<< "+ + + + + + + + + + + + + + +" <<std::endl;
    std::cout<< "decomp-EVE: " << evt_decomp[0] - ev_eve <<std::endl;
    std::cout<< "+ + + + + + + + + + + + + + +" <<std::endl;

    // for an elliptic object
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    unsigned Nx, Ny, p;
    double jfactor;
    std::cout << "Type p, Nx and Ny and jfactor (1.0)! \n";
    std::cin >> p >> Nx >> Ny; //more N means less iterations for same error
    std::cin >> jfactor;
    dg::Grid2d grid( 0, lx, 0, ly, p, Nx, Ny, bcx, bcy);
    dg::DVec w2d = dg::create::weights( grid);
    dg::DVec v2d = dg::create::inv_weights( grid);
    dg::DVec one = dg::evaluate( dg::one, grid);
    b = dg::evaluate( rhs, grid);
    dg::DVec chi =  dg::evaluate( pol, grid);
    dg::DVec chi_inv(chi);
    dg::blas1::transform( chi, chi_inv, dg::INVERT<double>());
    dg::blas1::pointwiseDot( chi_inv, v2d, chi_inv);
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> ell( grid, dg::not_normed, dg::centered, jfactor);
    ell.set_chi( chi);
    std::cout << "Created Polarisation object and set chi!\n";
    //   solve with decomposition
    dg::EVarbitraryMatrix ell_decomp(p*p*Nx*Ny);
    ell_decomp( ell, evt_decomp, evb_decomp);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via decomposition: " << evt_decomp[0] <<std::endl;
    //   solve with EVE
    x = dg::evaluate( initial, grid);
    dg::EVE<dg::DVec> ell_eve( x, p*p*Nx*Ny);
    double ev_max;
    niter = ell_eve( ell, x, b, ev_max);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via EVEs guess: " << ev_max << " after "<< niter << " iterations"<<std::endl;
    //    does it work with eInvert (tested as mere wrapper for EVE)
    b = dg::evaluate( rhs, grid);
    x = dg::evaluate( initial, grid);
    dg::eInvert<dg::DVec> invert( x, p*p*Nx*Ny, 1e-6, 1); //, false);
    niter = invert( ell, x, b, w2d, chi_inv, v2d, ev_eve);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "max. EV via eInvert guess: " << ev_eve << " after "<< niter << " iterations"<<std::endl;
    return 0;
}
