#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "blas.h"
#include "elliptic.h"
#include "eve.h"
#include "chebyshev.h"
#include "cg.h"
#include "backend/timer.cuh"
#include "manualmatrix.h"


/* Test maximum eigenvalue estimations by eve.h:
   - for elliptic via eInvert
   - for random matrix from manualmatrix.h       */

// Generate analytical test for elliptic
const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;

double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999; // the closer to one the worse for the solver
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);    //must be strictly positive
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}
double sol(double x, double y)
{   return sin( x)*sin(y);
}
double der(double x, double y)
{   return cos( x)*sin(y);
}


int main()
{   dg::Timer t;
  /* elliptic example */
    unsigned n, Nx, Ny;
    double eps, jfactor;
    std::cout << "Type n, Nx and Ny, epsilon, jfactor (1)! \n";
    std::cin >> n >> Nx >> Ny; //more N means less iterations for same error
    std::cin >> eps >> jfactor;
    std::cout << "Computation on: "<< n <<" x "<<Nx<<" x "<<Ny<<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly, n, Nx, Ny, bcx, bcy);
    dg::DVec w2d = dg::create::weights( grid);
    dg::DVec v2d = dg::create::inv_weights( grid);
    dg::DVec one = dg::evaluate( dg::one, grid);
    //create functions A(chi) x = b
    dg::DVec b =    dg::evaluate( rhs, grid);
    dg::DVec chi =  dg::evaluate( pol, grid);
    dg::DVec chi_inv(chi);
    dg::blas1::transform( chi, chi_inv, dg::INVERT<double>());
    dg::blas1::pointwiseDot( chi_inv, v2d, chi_inv);
    std::cout << "Create Polarisation object and set chi!\n";
    t.tic();
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> pol( grid, dg::not_normed, dg::centered, jfactor);
    pol.set_chi( chi);
    t.toc();
    std::cout << "Creation of polarisation object took: "<<t.diff()<<"s\n";

    //solve with pCG
    dg::DVec x_pcg = dg::evaluate( initial, grid);
    std::cout<< "solving with PC..." <<std::endl;
    {   t.tic();
        dg::Invert<dg::DVec > invert( x_pcg, n*n*Nx*Ny, eps);
        std::cout<< invert( pol, x_pcg, b, w2d, chi_inv, v2d) << " iterations"<<std::endl;
        t.toc();
        std::cout<< "...took " << t.diff() <<std::endl;
        std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
    }
    dg::DVec x_ecg = dg::evaluate( initial, grid);
    double eve_max;
    std::cout<< "solving with eCG..." <<std::endl;
    {   t.tic();
        dg::eInvert<dg::DVec > invert( x_ecg, n*n*Nx*Ny, eps);
        std::cout<< invert( pol, x_ecg, b, w2d, chi_inv, v2d, eve_max) << " iterations" <<std::endl;
        t.toc();
        std::cout << "...took " << t.diff() <<std::endl;
        std::cout<< "final EV estimate " << eve_max << std::endl;
        std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
    }
    /* random spd matrix */
    unsigned N;
    std::cout << "Type N of SPD Matrix \n";
    std::cin >> N;
    dg::RandPSDmatrix<dg::DVec> spd(N, 1.1);
    std::cout<< "Eigen max. EV " << spd.get_maxev() << std::endl;
    std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
    dg::DVec alpha(N), beta(N);
    for( uint i=0; i<N; ++i)
      { alpha[i] = 0.0;
        beta[i] = 1.0;
      }
    dg::EVE<dg::DVec> eve(alpha, N*N);
    eve(spd, alpha, beta, eve_max);
    std::cout<< "EVE max. EV " << eve_max << std::endl;
    std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
}
