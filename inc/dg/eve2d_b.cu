#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

//#include "backend/xspacelib.cuh"
#include <thrust/device_vector.h>
#include "blas.h"


#include "elliptic.h"
#include "eve.h"
#include "chebyshev.h"
#include "cg.h"
#include "backend/timer.cuh"

//NOTE: IF DEVICE=CPU THEN THE POLARISATION ASSEMBLY IS NOT PARALLEL AS IT IS NOW

//global relative error in L2 norm is O(h^P)
//as a rule of thumb with n=4 the true error is err = 1e-3 * eps as long as eps > 1e3*err

const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
//const double eps = 1e-3; //# of pcg iterations increases very much if
// eps << relativer Abstand der exakten Lösung zur Diskretisierung vom Sinus

double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);    //must be strictly positive
}
//double pol( double x, double y) {return 1.; }
//double pol( double x, double y) {return 1. + sin(x)*sin(y) + x; } //must be strictly positive

double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}
//double rhs( double x, double y) { return 2.*sin( x)*sin(y);}
//double rhs( double x, double y) { return 2.*sin(x)*sin(y)*(sin(x)*sin(y)+1)-sin(x)*sin(x)*cos(y)*cos(y)-cos(x)*cos(x)*sin(y)*sin(y)+(x*sin(x)-cos(x))*sin(y) + x*sin(x)*sin(y);}
double sol(double x, double y)
{   return sin( x)*sin(y);
}
double der(double x, double y)
{   return cos( x)*sin(y);
}


int main()
{   dg::Timer t;
    unsigned n, Nx, Ny;
    double eps, evmaxmul, evminmul;
    double jfactor;
    std::cout << "Type n, Nx and Ny, epsilon, jfactor (1) and ev_max and ev_min multiplier! \n";
    std::cin >> n >> Nx >> Ny; //more N means less iterations for same error
    std::cin >> eps >> jfactor;
    std::cin >> evmaxmul >> evminmul;
    std::cout << "Computation on: "<< n <<" x "<<Nx<<" x "<<Ny<<std::endl;
    //std::cout << "# of 2d cells                 "<< Nx*Ny <<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly, n, Nx, Ny, bcx, bcy);
    dg::DVec w2d = dg::create::weights( grid);
    dg::DVec v2d = dg::create::inv_weights( grid);
    dg::DVec one = dg::evaluate( dg::one, grid);
    //create functions A(chi) x = b
    dg::DVec b =    dg::evaluate( rhs, grid);
    dg::DVec chi =  dg::evaluate( pol, grid);
    dg::DVec chi_inv(chi);
    dg::blas1::transform( chi, chi_inv, dg::INVERT<double>());
    dg::blas1::pointwiseDot( chi_inv, v2d, chi_inv);
    std::cout << "Create Polarisation object and set chi!\n";
    t.tic();
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> pol( grid, dg::not_normed, dg::centered, jfactor);
    pol.set_chi( chi);
    t.toc();
    std::cout << "Creation of polarisation object took: "<<t.diff()<<"s\n";

    //solve with pCG
    dg::DVec x_pcg = dg::evaluate( initial, grid);
    std::cout<< "solving with PC..." <<std::endl;
    {   t.tic();
        dg::Invert<dg::DVec > invert( x_pcg, n*n*Nx*Ny, eps);
        std::cout<< invert( pol, x_pcg, b, w2d, chi_inv, v2d) << " iterations"<<std::endl;
        t.toc();
        std::cout<< "...took " << t.diff() <<std::endl;
        std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
    }
    dg::DVec x_ecg = dg::evaluate( initial, grid);
    double eve_max;
    std::cout<< "solving with eCG..." <<std::endl;
    {   t.tic();
        dg::eInvert<dg::DVec > invert( x_ecg, n*n*Nx*Ny, eps);
        std::cout<< invert( pol, x_ecg, b, w2d, chi_inv, v2d, eve_max) << " iterations" <<std::endl;
        t.toc();
        std::cout << "...took " << t.diff() <<std::endl;
        std::cout<< "final EV estimate " << eve_max << std::endl;
        std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
    }
    dg::DVec x_inter = dg::evaluate( initial, grid);
    double ev_max;
    std::cout<< "solving with eCG..." <<std::endl;
    {   t.tic();
        dg::eInvert<dg::DVec > invert( x_inter, 500, eps);
        std::cout<< invert( pol, x_inter, b, w2d, chi_inv, v2d, ev_max) << " iterations" <<std::endl;
        t.toc();
        std::cout << "...took " << t.diff() <<std::endl;
        std::cout<< "final EV estimate " << ev_max << std::endl;
        std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
    }
    dg::DVec x_cheb0 = x_inter;//dg::evaluate( initial, grid);
    std::cout<< "solving with Chebychev..." <<std::endl;
    {   t.tic();
      dg::cInvert<dg::DVec > invert( x_cheb0, 1000000, eps);
      //      cheb( A, x_cheb, b, ev_max, ev_min, eps, max_iter)
      std::cout<< invert( pol, x_cheb0, b, eve_max*evmaxmul, eve_max*evminmul, w2d, chi_inv, v2d) << " iterations" <<std::endl;
      t.toc();
      std::cout << "...took " << t.diff() <<std::endl;
      std::cout<< "final EV estimate " << ev_max << std::endl;
      std::cout<< "- - - - - - - - - - - - - - - - - - - -" <<std::endl;
    }




//    std::cout << "Create EVE "<<t.diff()<<"s\n";
//    dg::eInvert<dg::DVec > invert( x, 100, eps);
//    double ev_max;
//    std::cout << eps<<" ";
//    t.tic();
//    std::cout << " "<< invert( pol, x, b, w2d, chi_inv, v2d, ev_max) << std::endl;
//    std::cout << "EV " << ev_max << std::endl;
//    t.toc();
//    std::cout << "Took "<<t.diff()<<"s\n";
}
