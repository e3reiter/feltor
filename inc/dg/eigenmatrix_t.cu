#include "hip/hip_runtime.h"
/* test eigenmatrix.h for manualmatrix/FELTOR matrices */

#include <iostream>
#include "manualmatrix.h"
#include "eigenmatrix.h"
#include "elliptic.h"
#include "backend/timer.cuh"

// "difficult physics" for elliptic
const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);    //must be strictly positive
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}

int main()
{   dg::Timer t;
    // for some random symmetric positive matrix
    int n_rand, div;
    std::cout<< "n for random psd and divisor, please:" <<std::endl;
    std::cin>> n_rand >> div;
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "constructing random psd ..." <<std::endl;
    dg::RandPSDmatrix<dg::DVec> spd(n_rand, 1.1);
    dg::EVarbitraryMatrix<dg::DVec> am(n_rand, div);
    double ev_max;
    am( spd, ev_max);
    std::cout << "EV_max: "<< ev_max <<std::endl;
    // now for the elliptic object
    unsigned p, Nx, Ny;
    double eps, jfactor;
    std::cout<< "- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -" <<std::endl;
    std::cout << "Type p, Nx and Ny and epsilon and jfactor (1), new divisor, too! \n";
    std::cin >> p >> Nx >> Ny;
    std::cin >> eps >> jfactor >> div;
    dg::Grid2d grid( 0, lx, 0, ly, p, Nx, Ny, bcx, bcy);
    dg::DVec chi =  dg::evaluate( pol, grid);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout << "Create Polarisation object and set chi!\n";
    t.tic();
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> ell( grid, dg::not_normed, dg::centered, jfactor);
    ell.set_chi( chi);
    t.toc();
    std::cout << "Creation of polarisation object took: "<<t.diff()<<"s\n";
    dg::EVarbitraryMatrix<dg::DVec> ae( p*p*Nx*Ny, 100);
    ae( ell, ev_max);
    std::cout << "EV_max: "<< ev_max <<std::endl;
    return 0;
}
