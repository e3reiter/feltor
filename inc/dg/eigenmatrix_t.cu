#include "hip/hip_runtime.h"
/* test eigenmatrix.h for manualmatrix/FELTOR matrices */

#include <iostream>
#include "manualmatrix.h"
#include "eigenmatrix.h"
#include "elliptic.h"

// "difficult physics" for elliptic
const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);    //must be strictly positive
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}

template< class Vector>
void printvector( Vector& v)
{ for( uint i=0; i<v.size(); ++i)
  {   std::cout<<v[i]<<std::endl;
  }
  std::cout<<"- - - - - - - -"<<std::endl;
}

int main()
{   // for some random symmetric positive matrix
    int n_rand, nev_top, nev_bot;
    std::cout<< "n for random psd and number of EVtop/EVbot, please:" <<std::endl;
    std::cin>> n_rand >> nev_top >> nev_bot;
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout<< "constructing random psd ..." <<std::endl;
    dg::RandPSDmatrix<dg::DVec> spd(n_rand, 1.1);
    dg::EVarbitraryMatrix am(n_rand);
    dg::DVec ev_top(nev_top, 0.0), ev_bot(nev_bot, 0.0);
    am( spd, ev_top, ev_bot);
    printvector<dg::DVec> (ev_top);
    printvector<dg::DVec> (ev_bot);
    // now for the elliptic object
    unsigned p, Nx, Ny;
    double eps, jfactor;
    std::cout<< "- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -" <<std::endl;
    std::cout << "Type p, Nx and Ny and epsilon and jfactor (1), #EVtop/EVbot too! \n";
    std::cin >> p >> Nx >> Ny;
    std::cin >> eps >> jfactor >> nev_top >> nev_bot;
    dg::Grid2d grid( 0, lx, 0, ly, p, Nx, Ny, bcx, bcy);
    dg::DVec chi =  dg::evaluate( pol, grid);
    std::cout<< "- - - - - - - - - - - - - -" <<std::endl;
    std::cout << "Create Polarisation object and set chi!\n";
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> ell( grid, dg::not_normed, dg::centered, jfactor);
    ell.set_chi( chi);
    dg::EVarbitraryMatrix ae( p*p*Nx*Ny);
    dg::DVec evell_top(nev_top, 0.0), evell_bot(nev_bot, 0.0);
    ae( ell, evell_top, evell_bot);
    printvector<dg::DVec> (evell_top);
    printvector<dg::DVec> (evell_bot);
    return 0;
}
