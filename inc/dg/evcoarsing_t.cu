#include "hip/hip_runtime.h"
/* check Chebyshev as a linear solver */

#include <iostream>
#include "manualmatrix.h"
#include "eigenmatrix.h"
#include "eve.h"
#include "elliptic.h"
#include "chebyshev.h"
#include "cg.h"

// Imitate "difficult" physical problem
const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;
double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}

template< class Vector>
void printvector( Vector& v)
{ for( uint i=0; i<v.size(); ++i)
    {   std::cout<<v[i]<<std::endl;
    }
  std::cout<<"- - - - - - - -"<<std::endl;
}


int main()
{   unsigned p, Nx, Ny;
    double eps;
    double jfactor;
    std::cout << "Type p, Nx and Ny and epsilon and jfactor (1.0)! \n";
    std::cin >> p >> Nx >> Ny; //more N means less iterations for same error
    std::cin >> eps >> jfactor;
    std::cout << "Computation on: "<< p <<" x "<<Nx<<" x "<<Ny<<std::endl;
    //std::cout << "# of 2d cells                 "<< Nx*Ny <<std::endl;
    dg::Grid2d grid( 0, lx, 0, ly, p, Nx, Ny, bcx, bcy);
    dg::DVec w2d = dg::create::weights( grid);
    dg::DVec v2d = dg::create::inv_weights( grid);
    dg::DVec one = dg::evaluate( dg::one, grid);
    //create functions A(chi) x = b
    dg::DVec x =    dg::evaluate( initial, grid);
    dg::DVec b =    dg::evaluate( rhs, grid);
    dg::DVec chi =  dg::evaluate( pol, grid);
    dg::DVec chi_inv(chi);
    dg::blas1::transform( chi, chi_inv, dg::INVERT<double>());
    dg::blas1::pointwiseDot( chi_inv, v2d, chi_inv);
    dg::DVec x_pcg = x;
    std::cout << "Create Polarisation object and set chi!\n";

    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> pol( grid, dg::not_normed, dg::centered, jfactor);
    pol.set_chi( chi);
    dg::Invert<dg::DVec > invert( x_pcg, p*p*Nx*Ny, eps);
    std::cout << "invert in #iterations "<< invert( pol, x_pcg, b, w2d, chi_inv, v2d) <<std::endl;
    //get all EV by decomposition
    dg::EVarbitraryMatrix pol_decomp(p*p*Nx*Ny);
    dg::DVec ev_top(2, 0.0), ev_bot(2, 0.0);
    pol_decomp( pol, ev_top, ev_bot);
    printvector<dg::DVec> (ev_top);
    printvector<dg::DVec> (ev_bot);
    return 0;
