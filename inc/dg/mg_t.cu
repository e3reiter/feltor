#include "hip/hip_runtime.h"
#include <iostream>
#include <typeinfo>
#include <cmath>
#include "blas.h"
#include "elliptic.h"
#include "eve.h"
#include "chebyshev.h"
#include "cg.h"
#include "backend/timer.cuh"


template< class Vector>
void printvector( const std::string& message, Vector& v)
{   std::cout<< message << " is of typeid: " << typeid(v).name() <<std::endl;
    for( uint i=0; i<v.size(); ++i)
    {   std::cout<<v[i]<<std::endl;
    }
    std::cout<<"- - - - - - - -"<<std::endl;
}

/* Test problem construction */
const double lx = M_PI;
const double ly = 2.*M_PI;
dg::bc bcx = dg::DIR;
dg::bc bcy = dg::PER;

double initial( double x, double y)
{   return 0.;
}
double amp = 0.9999;
double pol( double x, double y)
{   return 1. + amp*sin(x)*sin(y);    //must be strictly positive
}
double rhs( double x, double y)
{   return 2.*sin(x)*sin(y)*(amp*sin(x)*sin(y)+1)-amp*sin(x)*sin(x)*cos(y)*cos(y)-amp*cos(x)*cos(x)*sin(y)*sin(y);
}
double sol(double x, double y)
{   return sin( x)*sin(y);
}
double der(double x, double y)
{   return cos( x)*sin(y);
}

/* create a number of grids, h, 2h, 4h, solve and compare solution & afford & maximum EV.

 */
int main()
{   return 0;
}
