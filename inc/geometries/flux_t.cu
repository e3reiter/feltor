#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"
#include "file/read_input.h"

#include "dg/backend/timer.cuh"
#include "orthogonal.h"
#include "curvilinear.h"
//#include "guenther.h"
#include "solovev.h"
#include "flux.h"
#include "dg/ds.h"
#include "init.h"

#include "file/nc_utilities.h"

using namespace dg::geo::solovev;

thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::Grid2d& g)
{
    thrust::host_vector<double> out(g.size());
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((0*g.n() + k)*g.Nx() + j)*g.n()+l];
    return out;
}

double sineX( double x, double y) {return sin(x)*sin(y);}
double cosineX( double x, double y) {return cos(x)*sin(y);}
double sineY( double x, double y) {return sin(x)*sin(y);}
double cosineY( double x, double y) {return sin(x)*cos(y);}
typedef dg::FieldAligned< dg::CurvilinearGrid3d<dg::HVec> , dg::IHMatrix, dg::HVec> DFA;
//typedef dg::FieldAligned< OrthogonalGrid3d<dg::HVec> , dg::IHMatrix, dg::HVec> DFA;

int main( int argc, char* argv[])
{
    std::cout << "Type n, Nx, Ny, Nz\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;   
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    //write parameters from file into variables
    GeomParameters gp(js);
    Psip psip( gp); 
    std::cout << "Psi min "<<psip(gp.R_0, 0)<<"\n";
    std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    std::cout << "Constructing flux grid ... \n";
    t.tic();
    MagneticField c( gp);
    dg::geo::FluxGenerator<Psip, PsipR, PsipZ, PsipRR, PsipRZ, PsipZZ, Ipol, IpolR, IpolZ>
        flux( c.psip, c.psipR, c.psipZ, c.psipRR, c.psipRZ, c.psipZZ, c.ipol, c.ipolR, c.ipolZ, psi_0, psi_1, gp.R_0, 0., 1);
    dg::CurvilinearGrid3d<dg::HVec> g3d(flux, n, Nx, Ny,Nz, dg::DIR);
    dg::CurvilinearGrid2d<dg::HVec> g2d = g3d.perp_grid();
    //OrthogonalGrid3d<dg::HVec> g3d(gp, psi_0, psi_1, n, Nx, Ny,Nz, dg::DIR);
    //OrthogonalGrid2d<dg::HVec> g2d = g3d.perp_grid();
    dg::Grid2d g2d_periodic(g2d.x0(), g2d.x1(), g2d.y0(), g2d.y1(), g2d.n(), g2d.Nx(), g2d.Ny()+1); 
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    int ncid;
    file::NC_Error_Handle err;
    err = nc_create( "flux.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[2];
    err = file::define_dimensions(  ncid, dim3d, g2d_periodic);
    int coordsID[2], onesID, defID, confID, volID, divBID;
    err = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 2, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 2, dim3d, &coordsID[1]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim3d, &onesID);
    err = nc_def_var( ncid, "d",    NC_DOUBLE, 2, dim3d, &defID);
    err = nc_def_var( ncid, "R", NC_DOUBLE, 2, dim3d, &confID);
    err = nc_def_var( ncid, "vol", NC_DOUBLE, 2, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( psip, g2d);
    //g.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g2d_periodic).data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.r()[i];
        Y[i] = g2d.z()[i];
    }

    dg::HVec temp0( g2d.size()), temp1(temp0);
    dg::HVec w3d = dg::create::weights( g2d);

    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g2d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g2d_periodic).data());
    //err = nc_put_var_double( ncid, coordsID[2], g.z().data());

    //compute and write deformation into netcdf
    dg::blas1::pointwiseDivide( g2d.g_xy(), g2d.g_xx(), temp0);
    const dg::HVec ones = dg::evaluate( dg::one, g2d);
    X=g2d.g_yy();
    err = nc_put_var_double( ncid, defID, periodify(X, g2d_periodic).data());
    //compute and write conformalratio into netcdf
    dg::blas1::pointwiseDivide( g2d.g_yy(), g2d.g_xx(), temp0);
    X=g2d.g_xx();
    err = nc_put_var_double( ncid, confID, periodify(X, g2d_periodic).data());

    std::cout << "Construction successful!\n";

    ////compute error in volume element
    //const dg::HVec f_ = g2d.f();
    //dg::blas1::pointwiseDot( g2d.g_xx(), g2d.g_yy(), temp0);
    //dg::blas1::pointwiseDot( g2d.g_xy(), g2d.g_xy(), temp1);
    //dg::blas1::axpby( 1., temp0, -1., temp1, temp0); //temp0=1/g = g^xx g^yy - g^xy^2
    //solovev::flux::FieldY fieldY(gp);
    //dg::HVec fby = dg::pullback( fieldY, g2d);//?
    //dg::blas1::pointwiseDot( f_, fby,fby);
//  //       dg::blas1::scal( fby, 2.*M_PI);

    //dg::blas1::pointwiseDot( fby, fby, temp1);
    //dg::blas1::axpby( 1., temp1, -1., temp0, temp0); ////temp0= g_xx g_yy - g_xy^2 - g
    //double error = sqrt( dg::blas2::dot( temp0, w3d, temp0)/dg::blas2::dot( temp1, w3d, temp1));
    //std::cout<< "Rel Error in Determinant is "<<error<<"\n";

    dg::blas1::pointwiseDot( g2d.g_xx(), g2d.g_yy(), temp0);
    dg::blas1::pointwiseDot( g2d.g_xy(), g2d.g_xy(), temp1);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>()); //temp0=1/sqrt(g) = sqrt(g^xx g^yy - g^xy^2)
    dg::blas1::pointwiseDivide( ones, temp0, temp0); //temp0=sqrt(g)
    X=temp0;
    err = nc_put_var_double( ncid, volID, periodify(X, g2d_periodic).data());
    dg::blas1::axpby( 1., temp0, -1., g2d.vol(), temp0); //temp0 = sqrt(g)-vol
    double error = sqrt(dg::blas2::dot( temp0, w3d, temp0)/dg::blas2::dot( g2d.vol(), w3d, g2d.vol()));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    //dg::blas1::pointwiseDivide(ones,fby,temp1); //=sqrt(g)
    //dg::blas1::axpby( 1., temp1, -1., g2d.vol(), temp0);
    //error=sqrt(dg::blas2::dot( temp0, w3d, temp0))/sqrt( dg::blas2::dot(g2d.vol(), w3d, g2d.vol()));
    //std::cout << "Rel Error of volume form is "<<error<<"\n";

    const dg::HVec vol = dg::create::volume( g3d);
    dg::HVec ones3d = dg::evaluate( dg::one, g3d);
    double volume = dg::blas1::dot( vol, ones3d);

    std::cout << "TEST VOLUME IS:\n";
    if( psi_0 < psi_1) gp.psipmax = psi_1, gp.psipmin = psi_0;
    else               gp.psipmax = psi_0, gp.psipmin = psi_1;
    dg::geo::Iris<Psip> iris( c.psip, gp.psipmin, gp.psipmax);
    //dg::CylindricalGrid3d<dg::HVec> g3d( gp.R_0 -2.*gp.a, gp.R_0 + 2*gp.a, -2*gp.a, 2*gp.a, 0, 2*M_PI, 3, 2200, 2200, 1, dg::PER, dg::PER, dg::PER);
    dg::CartesianGrid2d g2dC( gp.R_0 -2.0*gp.a, gp.R_0 + 2.0*gp.a, -2.0*gp.a,2.0*gp.a,1, 2e3, 2e3, dg::PER, dg::PER);
    dg::HVec vec  = dg::evaluate( iris, g2dC);
    dg::HVec R  = dg::evaluate( dg::cooX2d, g2dC);
    dg::HVec g2d_weights = dg::create::volume( g2dC);
    double volumeRZP = 2.*M_PI*dg::blas2::dot( vec, g2d_weights, R);
    std::cout << "volumeXYP is "<< volume<<std::endl;
    std::cout << "volumeRZP is "<< volumeRZP<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeRZP - volume)/volume<<std::endl;
    std::cout << "Note that the error might also come from the volume in RZP!\n"; //since integration of jacobian is fairly good probably

    ///////////////////////////TEST 3d grid//////////////////////////////////////
    //std::cout << "Start DS test!"<<std::endl;
    //const dg::HVec vol3d = dg::create::volume( g3d);
    //t.tic();
    //DFA fieldaligned( flux::Field( gp, g3d.x(), g3d.f_x()), g3d, gp.rk4eps, dg::NoLimiter()); 
    //dg::DS<DFA, dg::DMatrix, dg::HVec> ds( fieldaligned, flux::Field(gp, g3d.x(), g3d.f_x()), dg::normed, dg::centered);

    //
    //t.toc();
    //std::cout << "Construction took "<<t.diff()<<"s\n";
    //dg::HVec B = dg::pullback( solovev::InvB(gp), g3d), divB(B);
    //dg::HVec lnB = dg::pullback( solovev::LnB(gp), g3d), gradB(B);
    //dg::HVec gradLnB = dg::pullback( solovev::GradLnB(gp), g3d);
    //dg::blas1::pointwiseDivide( ones3d, B, B);
    //dg::HVec function = dg::pullback( solovev::FuncNeu(gp), g3d), derivative(function);
    //ds( function, derivative);

    //ds.centeredT( B, divB);
    //double norm =  sqrt( dg::blas2::dot(divB, vol3d, divB));
    //std::cout << "Divergence of B is "<<norm<<"\n";

    //ds.centered( lnB, gradB);
    //std::cout << "num. norm of gradLnB is "<<sqrt( dg::blas2::dot( gradB,vol3d, gradB))<<"\n";
    //norm = sqrt( dg::blas2::dot( gradLnB, vol3d, gradLnB) );
    //std::cout << "ana. norm of gradLnB is "<<norm<<"\n";
    //dg::blas1::axpby( 1., gradB, -1., gradLnB, gradLnB);
    //X = divB;
    //err = nc_put_var_double( ncid, divBID, periodify(X, g2d_periodic).data());
    //double norm2 = sqrt(dg::blas2::dot(gradLnB, vol3d,gradLnB));
    //std::cout << "rel. error of lnB is    "<<norm2/norm<<"\n";
    err = nc_close( ncid);



    return 0;
}
