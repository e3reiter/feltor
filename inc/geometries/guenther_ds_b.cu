#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/print.h>
#include <cusp/csr_matrix.h>
#include "file/read_input.h"
// #include "file/nc_utilities.h"

#include "dg/backend/xspacelib.cuh"
#include "dg/backend/evaluation.cuh"
#include "dg/backend/timer.cuh"
#include "dg/blas.h"
#include "dg/ds.h"
#include "dg/backend/functions.h"
#include "dg/functors.h"
#include "dg/elliptic.h"
#include "dg/cg.h"
// #include "draw/host_window.h"
#include "guenther.h"
#include "magnetic_field.h"
#include "testfunctors.h"

using namespace dg::geo::guenther;

int main( )
{

    /////////////////initialize params////////////////////////////////
    Json::Reader reader;
    Json::Value js;
    std::ifstream is("guenther_params.js");
    reader.parse(is,js,false);
    GeomParameters gp(js);
    gp.display( std::cout);

    //////////////////////////////////////////////////////////////////////////
    
    double Rmin=gp.R_0-1.0*gp.a;
    double Zmin=-1.0*gp.a*gp.elongation;
    double Rmax=gp.R_0+1.0*gp.a; 
    double Zmax=1.0*gp.a*gp.elongation;
    /////////////////////////////////////////////initialze fields /////////////////////
    
    Field field(gp.R_0, gp.I_0);
    InvB invb(gp.R_0, gp.I_0);
    GradLnB gradlnB(gp.R_0, gp.I_0);
    LnB lnB(gp.R_0, gp.I_0);
    FieldR bR_(gp.R_0, gp.I_0);
    FieldZ bZ_(gp.R_0, gp.I_0);
    FieldP bPhi_(gp.R_0, gp.I_0);
    FuncNeu funcNEU(gp.R_0,gp.I_0);
    FuncNeu2 funcNEU2(gp.R_0,gp.I_0);
    DeriNeu deriNEU(gp.R_0,gp.I_0);
    DeriNeu2 deriNEU2(gp.R_0,gp.I_0);
    DeriNeuT2 deriNEUT2(gp.R_0,gp.I_0);
    DeriNeuT deriNEUT(gp.R_0,gp.I_0);
    Divb divb(gp.R_0,gp.I_0);
    B Bfield(gp.R_0, gp.I_0);
    
    std::cout << "Type n, Nx, Ny, Nz\n";
    //std::cout << "Note, that function is resolved exactly in R,Z for n > 2\n";
    unsigned n=3, Nx=5, Ny=5, Nz=5;
    //std::cin >> n>> Nx>>Ny>>Nz;
    unsigned Nxn = Nx;
    unsigned Nyn = Ny;
    unsigned Nzn = Nz;

    double rk4eps = 1e-8;
    //std::cout << "Type RK4 eps (1e-8)\n";
    //std::cin >> rk4eps;
    double z0 = 0, z1 = 2.*M_PI;
    for (unsigned i=1;i<4;i+=2) { 

        Nzn = unsigned(Nz*pow(2,i));
        Nxn = (unsigned)ceil(Nx*pow(2,(double)(i*2./n)));
        Nyn = (unsigned)ceil(Ny*pow(2,(double)(i*2./n)));



        dg::CylindricalGrid3d<dg::DVec> g3d( Rmin,Rmax, Zmin,Zmax, z0, z1,  n,Nxn ,Nyn, Nzn,dg::DIR, dg::DIR, dg::PER);
        dg::Grid2d g2d( Rmin,Rmax, Zmin,Zmax,  n, Nxn ,Nyn);

        std::cout << "NR = " << Nxn << std::endl;
        std::cout << "NZ = " << Nyn<< std::endl;
        std::cout << "Nphi = " << Nzn << std::endl;
//            Nxn = (unsigned)ceil(Nxn*pow(2,(double)(2./n)));
//     Nyn = (unsigned)ceil( Nyn*pow(2,(double)(2./n)));

//        dg::Grid3d g3d( Rmin,Rmax, Zmin,Zmax, z0, z1,  n, Nx, Ny, Nz*pow(2,i),dg::DIR, dg::DIR, dg::PER);
//     dg::Grid2d g2d( Rmin,Rmax, Zmin,Zmax,  n, Nx, Ny); 
    const dg::DVec w3d = dg::create::volume( g3d);
    const dg::DVec w2d = dg::create::weights( g2d);
    const dg::DVec v3d = dg::create::inv_volume( g3d);

    std::cout << "computing dsDIR" << std::endl;
    dg::DDS::FieldAligned dsFA( field, g3d, rk4eps, dg::DefaultLimiter(), dg::DIR);
    std::cout << "computing dsNEU" << std::endl;
    dg::DDS::FieldAligned dsNUFA( field, g3d, rk4eps, dg::DefaultLimiter(), dg::NEU);

    dg::DDS ds ( dsFA, field, dg::not_normed, dg::centered), 
        dsNU ( dsNUFA, field, dg::not_normed, dg::centered);

//     dg::DS<dg::DMatrix, dg::DVec> dsNEU( field, g3d, g3d.hz(), rk4eps, dg::DefaultLimiter(), dg::NEU);
    
//     dg::Grid3d g3dp( Rmin,Rmax, Zmin,Zmax, z0, z1,  n, Nx, Ny, 1);
    
//     dg::DS<dg::DMatrix, dg::DVec> ds2d( field, g3dp, g3d.hz(), rk4eps, dg::DefaultLimiter(), dg::NEU);
    
    dg::DVec function = dg::evaluate( funcNEU, g3d) ,
                        temp( function),
                        temp2( function),
                        temp3( function),
                        derivative(function),
                        derivativeRZPhi(function),
                        diffRZPhi(function),
                        derivativef(function),
                        derivativeb(function),
                        derivativeones(function),
                        derivative2(function),
                        inverseB( dg::evaluate(invb, g3d)),
                        derivativeT(function),
                        logB( dg::evaluate(lnB, g3d)),
                        derivativeT2(function),
                        derivativeTones(function),
                        derivativeTds(function),
                        functionTinv(dg::evaluate( dg::zero, g3d)),
                        functionTinv2(dg::evaluate( dg::zero, g3d)),
                        dsTds(function),
                        dsTdsb(function),
                        dsTdsf(function),
                        dsTdsbd(function),
                        dsTdsfd(function),
                        dsTdsfb(function),
                        dsTdsfbd(function),
                        dsz(function),
                        divbsol(dg::evaluate(divb, g3d)),
                        divbT(function),
                        divBT(function),
                        lambda(function),
                        omega(function),
                        dsTds2(function);


    dg::DVec ones = dg::evaluate( dg::one, g3d);
    const dg::DVec function2 = dg::evaluate( funcNEU2, g3d);
    const dg::DVec solution = dg::evaluate( deriNEU, g3d);
    const dg::DVec solutionT = dg::evaluate( deriNEUT, g3d);
    const dg::DVec solutiondsz = dg::evaluate( deriNEU2, g3d);
    const dg::DVec solutiondsTds = dg::evaluate( deriNEUT2, g3d);

    const dg::DVec bhatR = dg::evaluate( bR_, g3d);
    const dg::DVec bhatZ = dg::evaluate( bZ_, g3d);
    const dg::DVec bhatPhi = dg::evaluate(bPhi_, g3d);
//     const dg::DVec Bfield_ = dg::evaluate(Bfield, g3d);
    const dg::DVec gradlnB_ = dg::evaluate(gradlnB, g3d);
//     dg::DMatrix dR(dg::create::dx( g3d, g3d.bcx(),dg::normed,dg::centered));
//     dg::DMatrix dZ(dg::create::dy( g3d, g3d.bcy(),dg::normed,dg::centered));
//     dg::DMatrix dphi(dg::create::ds( g3d, g3d.bcz(), dg::normed,dg::centered));
    
//     ds.set_boundaries( dg::PER, 0, 0);
    //direct gradpar method
//     dg::blas2::gemv( dR, function, temp); //d_R src
//     dg::blas2::gemv( dZ, function, temp2);  //d_Z src
//     dg::blas2::gemv( dphi, function, temp3);  //d_phi src
//     dg::blas1::pointwiseDot( bhatR, temp, temp); // b^R d_R src
//     dg::blas1::pointwiseDot( bhatZ, temp2, temp2); // b^Z d_Z src
//     dg::blas1::pointwiseDot( bhatPhi, temp3, temp3); // b^phi d_phi src
//     dg::blas1::axpby( 1., temp, 1., temp2 ); // b^R d_R src +  b^Z d_Z src
//     dg::blas1::axpby( 1., temp3, 1., temp2,derivativeRZPhi ); // b^R d_R src +  b^Z d_Z src + b^phi d_phi src
// 
//     dg::GeneralEllipticSym<dg::DMatrix, dg::DVec, dg::DVec> ellipticsym( g3d, dg::normed, dg::forward);
//     ellipticsym.set_x(bhatR);
//     ellipticsym.set_y(bhatZ );
//     ellipticsym.set_z(bhatPhi);
//     
//     
  
    dsNU( function, derivative); //ds(f)

//     dsNU.forward( function, derivativef); //ds(f)
//     dsNU.backward( function, derivativeb); //ds(f)

//     ds( ones, derivativeones); //ds(f)
//     ds( function2, derivative2); //ds(f)
//     //compute dsz
//     ds( inverseB, lambda); //gradpar 1/B
//     dg::blas1::pointwiseDivide(lambda,  inverseB, lambda); //-ds lnB
//     ds(function,omega); //ds T
//     dg::blas1::pointwiseDot(omega, lambda, omega);            //- ds lnB ds T
//     dg::blas1::pointwiseDot(omega, gradlnB_, omega);            //- ds lnB ds T
//     dg::blas1::axpby(1.0, omega, 0., dsz,dsz);    
    //     dg::blas1::axpby(-1.0, omega, 0., dsz,dsz);    

    
//     dsNU.forward(derivativeb,temp);
//     dsNU.backward(derivativef,omega);
//     dg::blas1::axpby( -1.0, omega, -0.0, temp,dsz);
//     ds( derivative, dsz); //ds(ds(f))

//     ds.dsz(function,dsz);       
//     dsNU( function, derivative); //ds(f)
//     dg::blas1::pointwiseDot(derivative, gradlnB_, omega);            //- ds lnB ds T
// 
//ds^2 T 
//     dg::blas1::axpby( -1.0, omega, 1.,dsz, dsTdsfbd);

//     
//     
//     ds.centeredT(function, derivativeT); //ds(f)
// 
//     //divB
//     dg::blas1::pointwiseDivide(ones,  inverseB, temp2); //B
//     ds.centeredT(temp2, divBT); // dsT B
// 
//     
//     ds.centeredT( function2, derivativeT2); //ds(f)
//     ds.centeredT( ones, derivativeTones); //ds(f)
    //B ds f/B
//     dg::blas1::pointwiseDot( inverseB, function, temp);
//     ds( temp, derivativeTds);
//     dg::blas1::pointwiseDot( derivativeTds, Bfield_, derivativeTds);
    //oder ds f - f dslnB
//     ds( function, derivativeTds);
//     dg::blas1::pointwiseDot(function,gradlnB_,temp);
//     dg::blas1::axpby(- 1.0, temp, 1., derivativeTds,derivativeTds);


    
    //     dg::blas1::pointwiseDivide( derivativeTds, inverseB, derivativeTds);
//     
//     ds.centeredT( derivative, dsTds); //dsT(ds(f))
//     
//     //overwrite with sym from adjoint dg
//     ellipticsym.symv(function,dsTds);
//     dg::blas1::scal(dsTds,-1.0);
// //     ds.centeredT(ones,divbT);
    ds.forwardT( derivativef, dsTdsf);  //dsT(ds(f))
//     ds.backwardT( derivativeb, dsTdsb); //dsT(ds(f))

//     //centered
//     ds.centeredTD(derivative,dsTdsfbd);
//     ds.forwardTD( derivativef, dsTdsfd); //dsT(ds(f))
//     ds.backwardTD( derivativeb, dsTdsbd); //dsT(ds(f))

//     //arithmetic average
//     dg::blas1::axpby(0.5,dsTdsb,0.5,dsTdsf,dsTdsfb);
//     dg::blas1::axpby(0.5,dsTdsbd,0.5,dsTdsfd,dsTdsfbd); 
    ds.symv(function,dsTdsfb);
    dg::blas1::pointwiseDot(v3d,dsTdsfb,dsTdsfb);
        //ds( function, temp);
        //dg::blas1::pointwiseDot( temp, inverseB, temp);
        //ds(temp, dsTdsfb);
        //dg::blas1::pointwiseDivide( dsTdsfb, inverseB, dsTdsfb);
//     ds.centeredT( derivative2, dsTds2); //dsT(ds(f))
//     dg::blas1::pointwiseDivide(ones,  inverseB, temp2); //B
     ds.centeredT( ones, divbT);
//     
//     double normdsds =dg::blas2::dot(derivative2, w3d,derivative2);
//     double normds1ds =dg::blas2::dot(derivativeones, w3d,derivative2);
//     double normdivBT =dg::blas2::dot(divBT, w3d,divBT);
     double normdivbT =dg::blas2::dot(divbT, w3d,divbT);
     double normdivb =dg::blas2::dot(divbsol, w3d,divbsol); 
//     double normdsTf = dg::blas2::dot(derivativeT2, w3d, function2);
//     double normdsT_1 = dg::blas2::dot(derivativeT2, w3d, ones);
//     double normdsT1 = dg::blas2::dot(derivativeTones, w3d, function2);
//     double normfds = dg::blas2::dot(function2, w3d, derivative2);
//     double norm1ds = dg::blas2::dot(ones, w3d, derivative2);
//     double normfdsTds = dg::blas2::dot(function2, w3d, dsTds2);
//     double norm1dsTds = dg::blas2::dot(ones, w3d, dsTds2);
//     
//     double norm1dsTB = dg::blas2::dot(ones, w3d, divBT);
//     double normBds1 = dg::blas2::dot(temp2, w3d, derivativeones);
//     double normfds1 = dg::blas2::dot(function2, w3d, derivativeones);
// 
    std::cout << "--------------------testing ds" << std::endl;
    double norm = dg::blas2::dot( w3d, solution);
    std::cout << "|| Solution ||   "<<sqrt( norm)<<"\n";
    double err =dg::blas2::dot( w3d, derivative);
    std::cout << "|| Derivative || "<<sqrt( err)<<"\n";
    dg::blas1::axpby( 1., solution, -1., derivative);
    err =dg::blas2::dot( w3d, derivative);
    std::cout << "Relative Difference in DS is "<< sqrt( err/norm )<<"\n"; 
   
//     std::cout << "--------------------testing ds with RZPhi method" << std::endl;
//     std::cout << "|| Solution ||   "<<sqrt( norm)<<"\n";
//     double errRZPhi =dg::blas2::dot( w3d, derivativeRZPhi);
//     std::cout << "|| Derivative || "<<sqrt( errRZPhi)<<"\n";
//     dg::blas1::axpby( 1., solution, -1., derivativeRZPhi);
//     errRZPhi =dg::blas2::dot( w3d, derivativeRZPhi);    
//     std::cout << "Relative Difference in DS is "<< sqrt( errRZPhi/norm )<<"\n"; 
//     
     std::cout << "--------------------testing dsT" << std::endl;
     std::cout << "|| divbsol ||  "<<sqrt( normdivb)<<"\n";
     std::cout << "|| divbT  ||   "<<sqrt( normdivbT)<<"\n";
     dg::blas1::axpby( 1., divbsol, -1., divbT);
     normdivbT =dg::blas2::dot(divbT, w3d,divbT);
     std::cout << "Relative Difference in DST is   "<<sqrt( normdivbT)<<"\n";
     std::cout << "-------------------- " << std::endl;
     //std::cout << "|| divB || "<<sqrt( normdivBT)<<"\n";
// 
//     
//     std::cout << "-------------------- " << std::endl;
//     double normT = dg::blas2::dot( w3d, solutionT);
//     std::cout << "|| SolutionT  ||  "<<sqrt( normT)<<"\n";
//     double errT =dg::blas2::dot( w3d, derivativeT);
//     std::cout << "|| DerivativeT || "<<sqrt( errT)<<"\n";
//     dg::blas1::axpby( 1., solutionT, -1., derivativeT);
//     errT =dg::blas2::dot( w3d, derivativeT);
//     std::cout << "Relative Difference in DST is "<< sqrt( errT/normT )<<"\n"; 
//     dg::blas1::axpby( 1., derivative, -1., derivativeT,omega);
//     double errTdiffdsdsT =dg::blas2::dot( w3d, omega);
//     std::cout << "Relative Difference in DST to DS is "<< sqrt( errTdiffdsdsT/norm )<<"\n";   
//     std::cout << "--------------------testing dsT with ds" << std::endl;
//     std::cout << "|| SolutionT ||     "<<sqrt( normT)<<"\n";
//     double errTds =dg::blas2::dot( w3d, derivativeTds);
//     std::cout << "|| DerivativeTds || "<<sqrt( errTds)<<"\n";
//     dg::blas1::axpby( 1., solutionT, -1., derivativeTds);
//     errTds =dg::blas2::dot( w3d, derivativeTds);
//     std::cout << "Relative Difference in DST is "<< sqrt( errTds/normT )<<"\n"; 
//     std::cout << "--------------------testing dsTds " << std::endl;
  
    double normdsTds = dg::blas2::dot( w3d, solutiondsTds);
//     std::cout << std::setprecision(16);
//     std::cout << "is the norm of the testfunction zero ? =       "<<sqrt( normdsTds)<<"\n";
//     for (unsigned i=0; i<g3d.size();i++){
//         std::cout << "solvalue " << solutiondsTds[i]<< std::endl;
//     }
//     double normnormdsTds = dg::blas2::dot(ones, w3d, solutiondsTds);
//     std::cout << "is the norm of the testfunction zero ? =       "<< normnormdsTds<<"\n";


//     std::cout << "|| SolutionT ||      "<<sqrt( normdsTds)<<"\n";
//     double errdsTds =dg::blas2::dot( w3d,dsTds);
//     std::cout << "|| DerivativeTds ||  "<<sqrt( errdsTds)<<"\n";
//     dg::blas1::axpby( 1., solutiondsTds, -1., dsTds);
//     errdsTds =dg::blas2::dot( w3d, dsTds);
//     std::cout << "Relative Difference in DST is "<< sqrt( errdsTds/normdsTds )<<"\n";   
    
    std::cout << "--------------------testing dsTdsfb " << std::endl;
    std::cout << "|| SolutionT ||      "<<sqrt( normdsTds)<<"\n";
    double remainder =dg::blas1::dot( w3d,dsTdsfb);
    double errdsTdsfb =dg::blas2::dot( w3d,dsTdsfb);
    std::cout << "|| DerivativeTds ||  "<<sqrt( errdsTdsfb)<<"\n";
    std::cout << "   Integral          "<<remainder<<"\n";
    dg::blas1::axpby( 1., solutiondsTds, -1., dsTdsfb);
    errdsTdsfb =dg::blas2::dot( w3d, dsTdsfb);
    std::cout << "Relative Difference in DST is "<< sqrt( errdsTdsfb/normdsTds )<<"\n";
//   
//     std::cout << "--------------------testing dsTdsfb with direct method" << std::endl;
//     std::cout << "|| SolutionT ||      "<<sqrt( normdsTds)<<"\n";
//     double errdsTdsfbd =dg::blas2::dot( w3d,dsTdsfbd);
//     std::cout << "|| DerivativeTds ||  "<<sqrt( errdsTdsfbd)<<"\n";
//     dg::blas1::axpby( 1., solutiondsTds, -1., dsTdsfbd);
//     errdsTdsfbd =dg::blas2::dot( w3d, dsTdsfbd);
//     std::cout << "Relative Difference in DST is "<< sqrt( errdsTdsfbd/normdsTds )<<"\n";
    

//     std::cout << "--------------------testing dsTds with dsz" << std::endl;
//     double normdsz = dg::blas2::dot( w3d, solutiondsz);
// 
//     std::cout << "|| Solution ||      "<<sqrt( normdsz)<<"\n";
//     double errdsz =dg::blas2::dot( w3d,dsz);
//     std::cout << "|| dsz ||  "<<sqrt( errdsz)<<"\n";
//     dg::blas1::axpby( 1., solutiondsz, -1., dsz);
//     errdsz =dg::blas2::dot( w3d, dsz);
//     std::cout << "Relative Difference in DST is "<< sqrt( errdsz/normdsz )<<"\n";   
//     
//     std::cout << "--------------------testing adjointness " << std::endl;
//     std::cout << "<f,ds(f)>   = "<< normfds<<"\n";
//     std::cout << "-<dsT(f),f> = "<< -normdsTf<<"\n";
//     std::cout << "Diff        = "<< normfds+normdsTf<<"\n";     
//     std::cout << "-------------------- " << std::endl;
// 
//     std::cout << "<B,ds(1)>   = "<< normBds1<<"\n";
//     std::cout << "-<dsT(B),1> = "<< -norm1dsTB<<"\n";
//     std::cout << "Diff        = "<< normBds1+norm1dsTB<<"\n";     
//     std::cout << "-------------------- " << std::endl;
//     
//     std::cout << "<f,ds(1)>   = "<< normfds1<<"\n";
//     std::cout << "-<dsT(f),1> = "<< -normdsT_1<<"\n";
//     std::cout << "Diff        = "<< normfds1+normdsT_1<<"\n";   
//     std::cout << "-------------------- " << std::endl;
//     
//     std::cout << "<1,ds(f)>   = "<< norm1ds<<"\n";
//     std::cout << "-<dsT(1),f> = "<< -normdsT1<<"\n";
//     std::cout << "Diff        = "<< norm1ds+normdsT1<<"\n";   
//     std::cout << "-------------------- " << std::endl;
//   
//     std::cout << "<f,dsT(ds(f))> = "<< normfdsTds<<"\n";
//     std::cout << "-<ds(f),ds(f)> = "<< -normdsds<<"\n";
//     std::cout << "Diff           = "<< normfdsTds+normdsds<<"\n";     
//     std::cout << "-------------------- " << std::endl;
//    
//     std::cout << "<1,dsT(ds(f))> = "<< norm1dsTds<<"\n";
//     std::cout << "-<ds(1),ds(f)> = "<< -normds1ds<<"\n";
//     std::cout << "Diff           = "<< norm1dsTds+normds1ds<<"\n";    
//     
// 
//     std::cout << "--------------------testing GeneralElliptic with inversion " << std::endl; 
//    //set up the parallel diffusion
//     dg::GeneralEllipticSym<dg::DMatrix, dg::DVec, dg::DVec> elliptic( g3d, dg::not_normed, dg::forward);
//     elliptic.set_x(bhatR);
//     elliptic.set_y(bhatZ );
//     elliptic.set_z(bhatPhi);
    
    
    double eps =1e-8;   
    dg::Invert< dg::DVec> invert( dg::evaluate(dg::zero,g3d), w3d.size(), eps );  
    std::cout << "MAX # iterations = " << w3d.size() << std::endl;
// 
//    const dg::DVec rhs = dg::evaluate( solovev::DeriNeuT2( gp.R_0, gp.I_0), g3d);
// // 
//     std::cout << " # of iterations "<< invert( elliptic, functionTinv, rhs ) << std::endl; //is dsTds 
//   
    double normf = dg::blas2::dot( w3d, function);
// 
//     std::cout << "Norm analytic Solution  "<<sqrt( normf)<<"\n";
//     double errinvT =dg::blas2::dot( w3d, functionTinv);
//     std::cout << "Norm numerical Solution "<<sqrt( errinvT)<<"\n";
// 
//     dg::blas1::axpby( 1., function, +1.,functionTinv);
//     errinvT =dg::blas2::dot( w3d, functionTinv);
//     std::cout << "Relative Difference is  "<< sqrt( errinvT/normf )<<"\n";
//     
    std::cout << "--------------------testing dsT" << std::endl; 
    std::cout << " # of iterations "<< invert( dsNU, functionTinv2,solutiondsTds ) << std::endl; //is dsTds
    std::cout << "Norm analytic Solution  "<<sqrt( normf)<<"\n";
    double errinvT2 =dg::blas2::dot( w3d, functionTinv2);
    std::cout << "Norm numerical Solution "<<sqrt( errinvT2)<<"\n";
    dg::blas1::axpby( 1., function, -1.,functionTinv2);
    errinvT2 =dg::blas2::dot( w3d, functionTinv2);
    std::cout << "Relative Difference is  "<< sqrt( errinvT2/normf )<<"\n";

//write netcdf
//     file::NC_Error_Handle err;
//     int ncid,tvarID;
//     err = nc_create( "out3.nc",NC_NETCDF4|NC_CLOBBER, &ncid);
//     dg::DVec transferD( dg::evaluate(dg::zero, g3d));
//     dg::HVec transferH( dg::evaluate(dg::zero, g3d));
// 
//     int dim_ids[4];
//     err = file::define_dimensions( ncid, dim_ids, &tvarID, g3d);
//     std::string names[3] = {"TG","TD","TA"}; 
//     int dataIDs[3]; 
//     size_t start[4] = {0, 0, 0, 0};
//     size_t count[4] = {1, g3d.Nz(), g3d.n()*g3d.Ny(), g3d.n()*g3d.Nx()};
//     err = nc_def_var( ncid, names[0].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[0]);  
//     err = nc_def_var( ncid, names[1].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[1]);  
//     err = nc_def_var( ncid, names[2].data(), NC_DOUBLE, 4, dim_ids, &dataIDs[2]);
//     err = nc_enddef( ncid);
//     err = nc_open("out3.nc", NC_WRITE, &ncid);
//     transferD=dsTds;
//     transferH =transferD;
//     err = nc_put_vara_double( ncid, dataIDs[0], start, count, transferH.data());
//     transferD=dsTdsfbd;
//     transferH =transferD;
//     err = nc_put_vara_double( ncid, dataIDs[1], start, count, transferH.data());    
//     transferD=dsTdsfb;
//     transferH =transferD;
//     err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data());    
//      err = nc_close(ncid);
    }
    
//     std::cout << "make Plot" << std::endl;
//     //make equidistant grid from dggrid
//     dg::HVec hvisual;
//     //allocate mem for visual
//     dg::HVec visual;
//     dg::HMatrix equigrid = dg::create::backscatter(g3d);               
// 
//     //evaluate on valzues from devicevector on equidistant visual hvisual vector
//     visual = dg::evaluate( dg::one, g3d);
//     //Create Window and set window title
//     GLFWwindow* w = draw::glfwInitAndCreateWindow( 100*Nz, 700, "");
//     draw::RenderHostData render(7 , 1*Nz);  
//     //create a colormap
//     draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
//     dg::DMatrix jump( dg::create::jump2d( g3d, g3d.bcx(), g3d.bcy(), dg::not_normed));
//     dg::blas2::symv( jump, ones, lambda);

//     std::stringstream title;
//     title << std::setprecision(10) << std::scientific;

//     while (!glfwWindowShouldClose( w ))
//     {
//         hvisual = divBT;
//         dg::blas2::gemv( equigrid, hvisual, visual);        
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"divB"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
// 
//         }
//         hvisual = derivativeT;         
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dsT(f)"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = derivative;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"ds(f)"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         dg::blas1::axpby(1.0,derivative,-1.0,derivativeT,omega);
//         hvisual = omega;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"diff"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = dsTds;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dsTdsfb"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = dsTds;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dsTds"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         hvisual = functionTinv2;
//         dg::blas2::gemv( equigrid, hvisual, visual);
//         colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), -100000000., thrust::maximum<double>()   );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );
//         title <<"dsz"<<" / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
//         for( unsigned k=0; k<Nz;k++)
//         {            
//             unsigned size=g3d.n()*g3d.n()*g3d.Nx()*g3d.Ny();            
//             dg::HVec part( visual.begin() + k*size, visual.begin()+(k+1)*size);
//             render.renderQuad( part, g3d.n()*g3d.Nx(), g3d.n()*g3d.Ny(), colors);
//         }
//         title << std::fixed; 
//         glfwSetWindowTitle(w,title.str().c_str());
//         title.str("");
//         glfwSwapBuffers(w);
//         glfwWaitEvents();
//     }
// 
//     glfwTerminate();
    return 0;
}
