#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <fstream>
#include <sstream>
#include <cmath>

#include "dg/backend/xspacelib.cuh"
#include "dg/functors.h"

#include "dg/backend/timer.cuh"
//#include "guenther.h"
#include "solovev.h"
#include "conformal.h"
#include "orthogonal.h"
#include "curvilinear.h"
#include "hector.h"
//#include "refined_conformal.h"
#include "dg/ds.h"
#include "init.h"

#include "file/nc_utilities.h"

using namespace dg::geo;

thrust::host_vector<double> periodify( const thrust::host_vector<double>& in, const dg::Grid2d& g)
{
    thrust::host_vector<double> out(g.size());
    for( unsigned i=0; i<g.Ny()-1; i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((i*g.n() + k)*g.Nx() + j)*g.n()+l];
    for( unsigned i=g.Ny()-1; i<g.Ny(); i++)
    for( unsigned k=0; k<g.n(); k++)
    for( unsigned j=0; j<g.Nx(); j++)
    for( unsigned l=0; l<g.n(); l++)
        out[((i*g.n() + k)*g.Nx() + j)*g.n()+l] = 
            in[((0*g.n() + k)*g.Nx() + j)*g.n()+l];
    return out;
}

int main( int argc, char* argv[])
{
    std::cout << "Type nHector, NxHector, NyHector ( 13 2 10)\n";
    unsigned nGrid, NxGrid, NyGrid;
    std::cin >> nGrid>> NxGrid>>NyGrid;   
    std::cout << "Type epsHector (1e-10)\n";
    double epsHector;
    std::cin >> epsHector;
    std::cout << "Type n, Nx, Ny, Nz ( 3 4 40 1)\n";
    unsigned n, Nx, Ny, Nz;
    std::cin >> n>> Nx>>Ny>>Nz;   
    Json::Reader reader;
    Json::Value js;
    if( argc==1)
    {
        std::ifstream is("geometry_params_Xpoint.js");
        reader.parse(is,js,false);
    }
    else
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    //write parameters from file into variables
    dg::geo::solovev::GeomParameters gp(js);
    dg::geo::solovev::Psip psip( gp); 
    std::cout << "Psi min "<<psip(gp.R_0, 0)<<"\n";
    std::cout << "Type psi_0 and psi_1\n";
    double psi_0, psi_1;
    std::cin >> psi_0>> psi_1;
    gp.display( std::cout);
    dg::Timer t;
    //solovev::detail::Fpsi fpsi( gp, -10);
    std::cout << "Constructing conformal grid ... \n";
    t.tic();
    dg::geo::solovev::MagneticField c( gp); 
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    Hector<dg::IDMatrix, dg::DMatrix, dg::DVec> hector( c.psip, c.psipR, c.psipZ, c.psipRR, c.psipRZ, c.psipZZ, psi_0, psi_1, gp.R_0, 0., nGrid, NxGrid, NyGrid, epsHector, true);
    dg::ConformalGrid3d<dg::HVec> g3d(hector, n, Nx, Ny,Nz, dg::DIR);
    dg::ConformalGrid2d<dg::HVec> g2d = g3d.perp_grid();
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //dg::geo::NablaPsiInvCollective<solovev::PsipR, solovev::PsipZ, solovev::PsipRR, solovev::PsipRZ, solovev::PsipZZ> nc( c.psipR, c.psipZ, c.psipRR, c.psipRZ, c.psipZZ);
    //dg::Hector<dg::IDMatrix, dg::DMatrix, dg::DVec> hector( c.psip, c.psipR, c.psipZ, c.psipRR, c.psipRZ, c.psipZZ, nc.nablaPsiInv, nc.nablaPsiInvX, nc.nablaPsiInvY, psi_0, psi_1, gp.R_0, 0., nGrid, NxGrid, NyGrid, epsHector, true);
    //dg::OrthogonalGrid3d<dg::HVec> g3d(hector, n, Nx, Ny,Nz, dg::DIR);
    //dg::OrthogonalGrid2d<dg::HVec> g2d = g3d.perp_grid();
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //dg::geo::LiseikinCollective<solovev::PsipR, solovev::PsipZ, solovev::PsipRR, solovev::PsipRZ, solovev::PsipZZ> lc( c.psipR, c.psipZ, c.psipRR, c.psipRZ, c.psipZZ, 0.1, 0.001);
    //dg::Hector<dg::IDMatrix, dg::DMatrix, dg::DVec> hector( c.psip, c.psipR, c.psipZ, c.psipRR, c.psipRZ, c.psipZZ, lc.chi_XX, lc.chi_XY, lc.chi_YY, lc.divChiX, lc.divChiY, psi_0, psi_1, gp.R_0, 0., nGrid, NxGrid, NyGrid, epsHector, true);
    //dg::CurvilinearGrid3d<dg::HVec> g3d(hector, n, Nx, Ny,Nz, dg::DIR);
    //dg::CurvilinearGrid2d<dg::HVec> g2d = g3d.perp_grid();
    //%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

    dg::Grid2d g2d_periodic(g2d.x0(), g2d.x1(), g2d.y0(), g2d.y1(), g2d.n(), g2d.Nx(), g2d.Ny()+1); 
    t.toc();
    std::cout << "Construction took "<<t.diff()<<"s"<<std::endl;
    std::cout << "Length in u is    "<<hector.width()<<std::endl;
    int ncid;
    file::NC_Error_Handle err;
    err = nc_create( "conformal.nc", NC_NETCDF4|NC_CLOBBER, &ncid);
    int dim3d[2];
    err = file::define_dimensions(  ncid, dim3d, g2d_periodic);
    int coordsID[2], onesID, defID, confID,volID,divBID;
    err = nc_def_var( ncid, "x_XYP", NC_DOUBLE, 2, dim3d, &coordsID[0]);
    err = nc_def_var( ncid, "y_XYP", NC_DOUBLE, 2, dim3d, &coordsID[1]);
    //err = nc_def_var( ncid, "z_XYP", NC_DOUBLE, 3, dim3d, &coordsID[2]);
    err = nc_def_var( ncid, "psi", NC_DOUBLE, 2, dim3d, &onesID);
    err = nc_def_var( ncid, "deformation", NC_DOUBLE, 2, dim3d, &defID);
    err = nc_def_var( ncid, "error", NC_DOUBLE, 2, dim3d, &confID);
    err = nc_def_var( ncid, "volume", NC_DOUBLE, 2, dim3d, &volID);
    err = nc_def_var( ncid, "divB", NC_DOUBLE, 2, dim3d, &divBID);

    thrust::host_vector<double> psi_p = dg::pullback( psip, g2d);
    //g.display();
    err = nc_put_var_double( ncid, onesID, periodify(psi_p, g2d_periodic).data());
    dg::HVec X( g2d.size()), Y(X); //P = dg::pullback( dg::coo3, g);
    for( unsigned i=0; i<g2d.size(); i++)
    {
        X[i] = g2d.r()[i];
        Y[i] = g2d.z()[i];
    }

    dg::HVec temp0( g2d.size()), temp1(temp0);
    dg::HVec w2d = dg::create::weights( g2d);

    err = nc_put_var_double( ncid, coordsID[0], periodify(X, g2d_periodic).data());
    err = nc_put_var_double( ncid, coordsID[1], periodify(Y, g2d_periodic).data());
    //err = nc_put_var_double( ncid, coordsID[2], g.z().data());

    //compute and write deformation into netcdf
    dg::blas1::pointwiseDivide( g2d.g_yy(), g2d.g_xx(), temp0);
    const dg::HVec ones = dg::evaluate( dg::one, g2d);
    X=temp0;
    err = nc_put_var_double( ncid, defID, periodify(X, g2d_periodic).data());
    //compute and write conformalratio into netcdf
    dg::blas1::pointwiseDivide( g2d.g_yy(), g2d.g_xx(), temp0);
    X=temp0;
    err = nc_put_var_double( ncid, confID, periodify(X, g2d_periodic).data());

    std::cout << "Construction successful!\n";

    //compare determinant vs volume form
    dg::blas1::pointwiseDot( g2d.g_xx(), g2d.g_yy(), temp0);
    dg::blas1::axpby( 1., temp0, -1., temp1, temp0);
    dg::blas1::transform( temp0, temp0, dg::SQRT<double>());
    dg::blas1::pointwiseDivide( ones, temp0, temp0);
    dg::blas1::transfer( temp0, X);
    err = nc_put_var_double( ncid, volID, periodify(X, g2d_periodic).data());
    dg::blas1::axpby( 1., temp0, -1., g2d.vol(), temp0);
    double error = sqrt(dg::blas2::dot( temp0, w2d, temp0)/dg::blas2::dot( g2d.vol(), w2d, g2d.vol()));
    std::cout << "Rel Consistency  of volume is "<<error<<"\n";

    std::cout << "TEST VOLUME IS:\n";
    dg::HVec vol = dg::create::volume( g2d);
    dg::HVec ones2d = dg::evaluate( dg::one, g2d);
    double volumeUV = dg::blas1::dot( vol, ones2d);

    vol = dg::create::volume( hector.internal_grid());
    ones2d = dg::evaluate( dg::one, hector.internal_grid());
    double volumeZE = dg::blas1::dot( vol, ones2d);
    std::cout << "volumeUV is "<< volumeUV<<std::endl;
    std::cout << "volumeZE is "<< volumeZE<<std::endl;
    std::cout << "relative difference in volume is "<<fabs(volumeUV - volumeZE)/volumeZE<<std::endl;
    err = nc_close( ncid);
    return 0;
}
