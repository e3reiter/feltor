#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/sparseblockmat.cuh"
#include "dg/backend/timer.cuh"
#include "dg/backend/average.cuh"
#include "dg/backend/typedefs.cuh"
#include "file/read_input.h"
#include "geometries/solovev.h"

#include "feltor.cuh"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the Feltor - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/
typedef dg::FieldAligned< dg::CylindricalGrid3d<dg::DVec>, dg::IDMatrix, dg::DVec> DFA;

int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v2,v3;
    std::stringstream title;
    if( argc == 1)
    {
        try{
            v = file::read_input("input.txt");
            v3 = file::read_input( "geometry_params.txt"); 
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else if( argc == 3)
    {
        try{
            v = file::read_input(argv[1]);
            v3 = file::read_input( argv[2]); 
        }catch( toefl::Message& m){
            m.display();
            return -1;
        }
    }
    else
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [geomfile] \n";
        return -1;
    }
    const eule::Parameters p( v);
    p.display( std::cout);
    const solovev::GeomParameters gp(v3);
    gp.display( std::cout);
    v2 = file::read_input( "window_params.txt");
    GLFWwindow* w = draw::glfwInitAndCreateWindow( (p.Nz+1)/v2[2]*v2[3], v2[1]*v2[4], "");
    draw::RenderHostData render(v2[1], (p.Nz+1)/v2[2]);



    //////////////////////////////////////////////////////////////////////////
    double Rmin=gp.R_0-p.boxscaleRm*gp.a;
    double Zmin=-p.boxscaleZm*gp.a*gp.elongation;
    double Rmax=gp.R_0+p.boxscaleRp*gp.a; 
    double Zmax=p.boxscaleZp*gp.a*gp.elongation;
    //Make grid
    dg::CylindricalGrid3d<dg::DVec> grid( Rmin,Rmax, Zmin,Zmax, 0, 2.*M_PI, p.n, p.Nx, p.Ny, p.Nz, p.bc, p.bc, dg::PER);  

    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::CylindricalGrid3d<dg::DVec>, dg::DS<DFA, dg::DMatrix, dg::DVec>, dg::DMatrix, dg::DVec> feltor( grid, p, gp); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::CylindricalGrid3d<dg::DVec>, dg::DS<DFA, dg::DMatrix, dg::DVec>, dg::DMatrix, dg::DVec> rolkar( grid, p, gp, feltor.ds(), feltor.dsDIR());
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    //background profile
    solovev::Nprofile prof(p.bgprofamp, p.nprofileamp, gp); //initial background profile
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); 
    //perturbation 
    dg::GaussianZ gaussianZ( 0., p.sigma_z*M_PI, 1); //modulation along fieldline
    if( p.mode == 0 || p.mode == 1)
    {
        dg::Gaussian init0( gp.R_0+p.posX*gp.a, p.posY*gp.a, p.sigma, p.sigma, p.amp);
        if( p.mode == 0)
            y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 3); //rounds =3 ->2*3-1
        if( p.mode == 1)
            y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); //rounds =1 ->2*1-1
    }
    if( p.mode == 2)
    {
        dg::BathRZ init0(16,16,p.Nz,Rmin,Zmin, 30.,5.,p.amp);
        y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); 
    }
    if( p.mode == 3)
    {
        solovev::ZonalFlow init0(p.amp, p.k_psi, gp);
        y1[1] = feltor.ds().fieldaligned().evaluate( init0, gaussianZ, (unsigned)p.Nz/2, 1); 
    }
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //sum up background and perturbation
    dg::blas1::plus(y0[1], -1); //initialize ni-1
    if( p.mode == 2 || p.mode == 3)
    {
        dg::DVec damping = dg::evaluate( solovev::GaussianProfXDamping( gp), grid);
        dg::blas1::pointwiseDot(damping,y0[1], y0[1]); //damp with gaussprofdamp
    }
    std::cout << "intiialize ne" << std::endl;
    if( p.initcond == 0) feltor.initializene( y0[1], y0[0]);
    if( p.initcond == 1) dg::blas1::axpby( 1., y0[1], 0.,y0[0], y0[0]); //set n_e = N_i
    std::cout << "Done!\n";

    dg::blas1::axpby( 0., y0[2], 0., y0[2]); //set Ue = 0
    dg::blas1::axpby( 0., y0[3], 0., y0[3]); //set Ui = 0

    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    std::cout << "intiialize karniadakis" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    std::cout << "Done!\n";
//     feltor.energies( y0);//now energies and potential are at time 0

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    dg::ToroidalAverage<dg::HVec> toravg(grid);
    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0., dEdt = 0.;
    
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
     //probe
    const dg::HVec Xprobe(1,gp.R_0+p.boxscaleRp*gp.a);
    const dg::HVec Zprobe(1,0.);
    const dg::HVec Phiprobe(1,M_PI);
    dg::IDMatrix probeinterp;
    dg::blas2::transfer( dg::create::interpolation( Xprobe, Zprobe, Phiprobe, grid, dg::NEU), probeinterp);
    dg::DVec probevalue(1,0.);   
    while ( !glfwWindowShouldClose( w ))
    {

        dg::blas1::transfer( y0[0], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() = 1.0;
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ne / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ne-1 / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);   
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw ions
        //thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-0.));//ne-1
        dg::blas1::transfer( y0[1], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
        colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ni / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ni-1 / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        //draw potential
        //transform to Vor
        dvisual=feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
        dg::blas1::transfer( y1[1], hvisual);
//         hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(),visual.end(), 0.,thrust::maximum<double>()  );
//         colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Phi / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        title <<"Omega / "<< colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        //draw U_e
        dg::blas1::transfer( y0[2], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0.,thrust::maximum<double>()  );
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Ue / "<<colors.scalemin()<<"  " << colors.scalemax()<<"\t";
        title <<"Ue / " << colors.scalemax()<<"\t";
                for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);      
        
        //draw U_i
        dg::blas1::transfer( y0[3], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), 0., thrust::maximum<double>()  );
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        colors.scalemin() = -colors.scalemax();
        //title <<"Ui / "<<colors.scalemin()<< "  " << colors.scalemax()<<"\t";
        title <<"Ui / " << colors.scalemax()<<"\t";
        for( unsigned k=0; k<p.Nz/v2[2];k++)
        {
            unsigned size=grid.n()*grid.n()*grid.Nx()*grid.Ny();
            dg::HVec part( visual.begin() + k*v2[2]*size, visual.begin()+(k*v2[2]+1)*size);
            render.renderQuad( part, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        }
        dg::blas1::axpby(0.0,avisual,0.0,avisual);
        toravg(visual,avisual);
        render.renderQuad( avisual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
//             feltor.energies( y0); //update energetics
            //Compute probe values
            dg::blas2::gemv(probeinterp,y0[0],probevalue);
            std::cout << " Ne_p - 1  = " << probevalue[0] <<"\t";
            dg::blas2::gemv(probeinterp,feltor.potential()[0],probevalue);
            std::cout << " Phi_p = " << probevalue[0] <<"\t";
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass0<<"\t";
            E1 = feltor.energy();
            dEdt = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << "Accuracy: "<< 2.*fabs((dEdt-diss)/(dEdt+diss))<<" d E/dt = " << dEdt <<" Lambda =" << diss << "\n";
            
            E0 = E1;

        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
