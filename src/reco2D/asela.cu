#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>

#include "draw/host_window.h"
//#include "draw/device_window.cuh"

#include "asela.cuh"

#include "dg/runge_kutta.h"
#include "dg/multistep.h"
#include "dg/backend/timer.cuh"
#include "parameters.h"

/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/

double aparallel( double x, double y)
{
    return 0.1/cosh(x)/cosh(x)*cos(1./8.*y);
}

int main( int argc, char* argv[])
{
    ////Parameter initialisation ////////////////////////////////////////////
    Json::Reader reader;
    Json::Value js;
    if( argc == 1)
    {
        std::ifstream is("input.json");
        reader.parse(is,js,false);
    }
    else if( argc == 2)
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    const eule::Parameters p( js);
    p.display( std::cout);
    /////////glfw initialisation ////////////////////////////////////////////
    std::stringstream title;
    std::ifstream is( "window_params.js");
    reader.parse( is, js, false);
    is.close();
    GLFWwindow* w = draw::glfwInitAndCreateWindow( js["width"].asDouble(), js["height"].asDouble(), "");
    draw::RenderHostData render(js["rows"].asDouble(), js["cols"].asDouble());
    /////////////////////////////////////////////////////////////////////////

    dg::Grid2d grid( -p.lxhalf, p.lxhalf, -p.lyhalf, p.lyhalf , p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    //create RHS 
    eule::Asela< dg::CartesianGrid2d, dg::DMatrix, dg::DVec > asela( grid, p); 
    eule::Diffusion<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> diffusion( grid, p.nu, 1., 1. );
    //create initial vector
    std::vector<dg::DVec> y0(4, dg::evaluate( dg::one, grid)), y1(y0); // n_e' = gaussian
    y0[2] = y0[3] = dg::evaluate( aparallel, grid);
    dg::DVec temp( y0[2]);
    dg::Elliptic<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> laplaceM(grid, dg::normed, dg::centered);
    dg::blas2::gemv( laplaceM, y0[2], temp); //u_e = \Delta A_parallel
    dg::blas1::axpby( p.dhat[0]*p.dhat[0], temp, 1., y0[2]);//w_e = \Delta A + beta/mue A
   
    for( unsigned i=0; i<2; i++)
        dg::blas1::transform( y0[i], y0[i], dg::LN<double>());

    dg::Karniadakis< std::vector<dg::DVec> > ab( y0, y0[0].size(), p.eps_time);

    dg::DVec dvisual( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExt colors( 1.);
    //create timer
    dg::Timer t;
    double time = 0;
    //ab.init( asela, y0, p.dt);
    ab.init( asela, diffusion, y0, p.dt);
    //ab( asela, y0, y1, p.dt);
    //y0.swap( y1); 
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    unsigned step = 0;
    while ( !glfwWindowShouldClose( w ))
    {
        for( unsigned i=0; i<2; i++)
            dg::blas1::transform( y0[i], y0[i], dg::EXP<double>());

        thrust::transform( y1[0].begin(), y1[0].end(), dvisual.begin(), dg::PLUS<double>(-1));
        dg::blas1::transfer(dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ne / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


        thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-1));
        dg::blas1::transfer(dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw ions
        title << std::setprecision(2) << std::scientific;
        title <<"ni / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


        dvisual = asela.potential()[0];
        dg::blas1::transfer(dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"Potential / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


        //transform phi
        dg::blas2::gemv(laplaceM, asela.potential()[0], dvisual);
        dg::blas1::transfer(dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"omega / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


        //transform Aparallel
        dvisual = asela.aparallel();
        dg::blas1::transfer(dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"Aparallel / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


        //transform Aparallel
        dg::blas2::gemv( laplaceM, asela.aparallel(), dvisual);
        dg::blas1::transfer(dvisual, hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        //compute the color scale
        colors.scale() =  (float)thrust::reduce( visual.begin(), visual.end(), 0., dg::AbsMax<double>() );
        //draw phi and swap buffers
        title <<"Jpar / "<<colors.scale()<<"\t";
        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        

        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            step++;
            try{ ab( asela, diffusion, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            //y0.swap( y1); //attention on -O3 ?
        }
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
