#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>

#include "dg/backend/timer.cuh"
#include "dg/algorithm.h"
#include "dg/functors.h"
#include "dg/backend/evaluation.cuh"
#include "dg/runge_kutta.h"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/typedefs.cuh"

#include "shu.cuh"
#include "parameters.h"



using namespace std;
using namespace dg;
//const unsigned k=4;
const double Tmax = 0.01;
const double eps = 1e-14;
const unsigned n=1; //make error in space small
unsigned Nx = 100, Ny = Nx;

int main( int argc, char * argv[])
{


    double dt0; 
    std::cout << "type dt0 (1e-3)!\n";
    std::cin >> dt0;
    std::cout << "k n dt Nx eps vort enstr energy\n";
    Grid2d grid( 0, 1, 0, 1, n, Nx, Ny, dg::PER, dg::PER);
    DVec w2d( create::weights(grid));
    dg::Lamb lamb( 0.5, 0.8, 0.1, 1.);
    const HVec omega = evaluate ( lamb, grid);
    Shu<dg::DMatrix, dg::DVec> shu( grid, eps);
    const DVec stencil = evaluate( one, grid);
    for(unsigned i=0; i<6;i++)
    {
        double dt = dt0/pow(2,i);
        unsigned NT = (unsigned)(Tmax/dt);
        //initiate solver 
        DVec y0( omega ), y1( y0);
        //make solver and stepper
        AB< 1, DVec > ab( y0);
        ab.init( shu, y0, dt);
        ab( shu, y1);

        double vorticity = blas2::dot( stencil, w2d, y1);
        double enstrophy = 0.5*blas2::dot( y1, w2d, y1);
        double energy =    0.5*blas2::dot( y1, w2d, shu.potential()) ;
        /////////////////////////////////////////////////////////////////
        try{
        for( unsigned i=0; i<NT; i++)
        {
            ab( shu, y1);
        }
        }
        catch( dg::Fail& fail) { 
            std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
            std::cerr << "Does Simulation respect CFL condition?\n";
        }
        std::cout << 1 <<" "<<n<<" "<<dt<<" "<<Nx<<" "<<eps<<" ";
        std::cout << fabs(blas2::dot( stencil , w2d, y1));
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, y1)-enstrophy)/enstrophy;
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, shu.potential())-energy)/energy <<"\n";
    }
    std::cout << std::endl;
    for(unsigned i=0; i<6;i++)
    {
        double dt = dt0/pow(2,i);
        unsigned NT = (unsigned)(Tmax/dt);
        //initiate solver 
        DVec y0( omega ), y1( y0);
        //make solver and stepper
        AB< 2, DVec > ab( y0);
        ab.init( shu, y0, dt);
        ab( shu, y1);

        double vorticity = blas2::dot( stencil, w2d, y1);
        double enstrophy = 0.5*blas2::dot( y1, w2d, y1);
        double energy =    0.5*blas2::dot( y1, w2d, shu.potential()) ;
        /////////////////////////////////////////////////////////////////
        try{
        for( unsigned i=0; i<NT; i++)
        {
            ab( shu, y1);
        }
        }
        catch( dg::Fail& fail) { 
            std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
            std::cerr << "Does Simulation respect CFL condition?\n";
        }
        std::cout << 2 <<" "<<n<<" "<<dt<<" "<<Nx<<" "<<eps<<" ";
        std::cout << fabs(blas2::dot( stencil , w2d, y1));
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, y1)-enstrophy)/enstrophy;
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, shu.potential())-energy)/energy <<"\n";
    }
    std::cout << std::endl;
    for(unsigned i=0; i<6;i++)
    {
        double dt = dt0/pow(2,i);
        unsigned NT = (unsigned)(Tmax/dt);
        //initiate solver 
        DVec y0( omega ), y1( y0);
        //make solver and stepper
        AB< 3, DVec > ab( y0);
        ab.init( shu, y0, dt);
        ab( shu, y1);

        double vorticity = blas2::dot( stencil, w2d, y1);
        double enstrophy = 0.5*blas2::dot( y1, w2d, y1);
        double energy =    0.5*blas2::dot( y1, w2d, shu.potential()) ;
        /////////////////////////////////////////////////////////////////
        try{
        for( unsigned i=0; i<NT; i++)
        {
            ab( shu,y1);
        }
        }
        catch( dg::Fail& fail) { 
            std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
            std::cerr << "Does Simulation respect CFL condition?\n";
        }
        std::cout << 3 <<" "<<n<<" "<<dt<<" "<<Nx<<" "<<eps<<" ";
        std::cout << fabs(blas2::dot( stencil , w2d, y1));
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, y1)-enstrophy)/enstrophy;
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, shu.potential())-energy)/energy <<"\n";
    }
    std::cout << std::endl;
    for(unsigned i=0; i<6;i++)
    {
        double dt = dt0/pow(2,i);
        unsigned NT = (unsigned)(Tmax/dt);
        //initiate solver 
        DVec y0( omega ), y1( y0);
        //make solver and stepper
        AB< 4, DVec > ab( y0);
        ab.init( shu, y0, dt);
        ab( shu, y1);

        double vorticity = blas2::dot( stencil, w2d, y1);
        double enstrophy = 0.5*blas2::dot( y1, w2d, y1);
        double energy =    0.5*blas2::dot( y1, w2d, shu.potential()) ;
        /////////////////////////////////////////////////////////////////
        try{
        for( unsigned i=0; i<NT; i++)
        {
            ab( shu, y1);
        }
        }
        catch( dg::Fail& fail) { 
            std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
            std::cerr << "Does Simulation respect CFL condition?\n";
        }
        std::cout << 4 <<" "<<n<<" "<<dt<<" "<<Nx<<" "<<eps<<" ";
        std::cout << fabs(blas2::dot( stencil , w2d, y1));
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, y1)-enstrophy)/enstrophy;
        std::cout << " "<<fabs(0.5*blas2::dot( y1, w2d, shu.potential())-energy)/energy <<"\n";
    }
    std::cout << std::endl;
    return 0;

}
