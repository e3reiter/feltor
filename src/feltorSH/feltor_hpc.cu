#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"

#include "dg/backend/interpolation.cuh"
#include "file/read_input.h"
#include "file/nc_utilities.h"

#include "feltor.cuh"
#include "parameters.h"


/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v3;
    std::string input;
    if( argc != 3)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        input = file::read_file( argv[1]); //deprecated, better use json reader directly, instead!
    }
    Json::Reader reader;
    Json::Value js;
    reader.parse( input, js, false);
    std::cout << js<<std::endl;
    input = js.toStyledString(); //save input without comments, which is important if netcdf file is later read by another parser
    const eule::Parameters p( js);
    p.display( std::cout);

      //Make grid
     dg::Grid2d grid( 0., p.lx, 0.,p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
     dg::Grid2d grid_out( 0., p.lx, 0.,p.ly, p.n_out, p.Nx_out, p.Ny_out, p.bc_x, p.bc_y);  
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> feltor( grid, p); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> rolkar( grid, p);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
       //initial perturbation
    dg::Gaussian init0( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
    dg::CONSTANT prof(p.bgprofamp );
    std::vector<dg::DVec> y0(4, dg::evaluate( prof, grid)), y1(y0); //Ne,Ni,Te,Ti = prof    
   
   //initialization via N_i,T_I ->n_e, t_i=t_e
    y1[1] = dg::evaluate( init0, grid);
    dg::blas1::pointwiseDot(y1[1], y0[1],y1[1]); //<n>*ntilde    
    dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize Ni = <n> + <n>*ntilde
    if (p.iso == 1) dg::blas1::axpby( 1.,y1[2], 0., y0[3]); //initialize Ti = prof
    if (p.iso == 0) dg::blas1::axpby( 1.,y0[1], 0., y0[3]); //initialize Ti = N_i
    dg::blas1::plus(y0[1], (-(p.bgprofamp + p.nprofileamp))); //= Ni - bg
    std::cout << "intiialize ne" << std::endl;
    if( p.init == 0)
        feltor.initializene( y0[1],y0[3], y0[0]);    //ne -bg
    else  
        dg::blas1::axpby( 1., y0[1], 0., y0[0], y0[0]); // for Omega*=0
    std::cout << "Done!\n";    
    
    std::cout << "intialize ti=te" << std::endl;
    if (p.iso == 1) {
        dg::blas1::transform(y0[3], y0[3], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =Ti - bg
        dg::blas1::axpby( 1.,y0[3], 0., y0[2]); //initialize Ti = N_i
    }
    if (p.iso == 0) {
        dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(+(p.bgprofamp + p.nprofileamp))); //Ni
        dg::blas1::pointwiseDot(y0[1],y0[3],y1[3]); // = Ni Ti
        dg::blas1::transform(y1[3], y1[3], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp)*(p.bgprofamp + p.nprofileamp))); //Pi = Pi - bg^2

        if( p.init == 0)
            feltor.initializepi(y1[3],y0[3], y0[2]); // = pi-bg^2    
        //compute ti-bg = ((pi-bg^2) +bg^2)/ne -bg
        dg::blas1::transform(y0[2], y0[2], dg::PLUS<>(+(p.bgprofamp + p.nprofileamp)*(p.bgprofamp + p.nprofileamp)));
        dg::blas1::transform(y0[0], y0[0], dg::PLUS<>(+(p.bgprofamp + p.nprofileamp))); //=ne    
        dg::blas1::pointwiseDivide(y0[2],y0[0],y0[2]);

        if( p.init != 0)
            dg::blas1::axpby( 1., y0[3], 0., y0[2], y0[2]); //for Omega*=0

        dg::blas1::transform(y0[2], y0[2], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp)));
        dg::blas1::transform(y0[0], y0[0], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =ne-bg
        dg::blas1::transform(y0[3], y0[3], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =Ti - bg
        dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); // =Ni - bg 
    }
    std::cout << "Done!\n";
   

    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[2],NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    const int version[3] = {FELTOR_MAJOR_VERSION, FELTOR_MINOR_VERSION, FELTOR_SUBMINOR_VERSION};
    err = nc_put_att_int( ncid, NC_GLOBAL, "feltor_major_version", NC_INT, 1, &version[0]);
    err = nc_put_att_int( ncid, NC_GLOBAL, "feltor_minor_version", NC_INT, 1, &version[1]);
    err = nc_put_att_int( ncid, NC_GLOBAL, "feltor_subminor_version", NC_INT, 1, &version[2]);
    int dim_ids[3], tvarID;
    err = file::define_dimensions( ncid, dim_ids, &tvarID, grid_out);
    err = nc_enddef( ncid);
    err = nc_redef(ncid);

    //field IDs
    std::string names[6] = {"electrons", "ions", "Telectrons","Tions", "potential","vor"}; 
    int dataIDs[6]; 
    for( unsigned i=0; i<6; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 3, dim_ids, &dataIDs[i]);}

    //energy IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, energyIDs[3], dissID, dEdtID, accuracyID;
    err = nc_def_var( ncid, "energy",   NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var( ncid, "mass",   NC_DOUBLE, 1, &EtimeID, &massID);
    std::string energies[3] = {"Se", "Si", "Uperp"}; 
    for( unsigned i=0; i<3; i++){
        err = nc_def_var( ncid, energies[i].data(), NC_DOUBLE, 1, &EtimeID, &energyIDs[i]);}
    err = nc_def_var( ncid, "dissipation",   NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var( ncid, "dEdt",     NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_def_var( ncid, "accuracy", NC_DOUBLE, 1, &EtimeID, &accuracyID);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    std::cout << "First output ... \n";
    size_t start[3] = {0, 0, 0};
    size_t count[3] = {1, grid_out.n()*grid_out.Ny(), grid_out.n()*grid_out.Nx()};
    dg::DVec transfer(  dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::IDMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    for( unsigned i=0; i<4; i++)
    {
        dg::blas2::gemv( interpolate, y0[i], transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    //pot
    transfer = feltor.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    dg::blas1::transfer( transferD, transferH);

    err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
    //Vor
    transfer = feltor.potential()[0];
    dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);            
    dg::blas2::symv( interpolate,y1[1], transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[5], start, count, transferH.data() );
    double time = 0;

    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);

    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    double energy0 = feltor.energy(), mass0 = feltor.mass(), E0 = energy0, mass = mass0, E1 = 0.0, dEdt = 0., diss = 0., accuracy=0.;


    std::vector<double> evec = feltor.energy_vector();
    err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &energy0);
    err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass0);
    for( unsigned i=0; i<3; i++)
        err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);

    err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
    err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
    //probe
    err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
    

    
    err = nc_close(ncid);
    std::cout << "First write successful!\n";

    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                return -1;
            }
            step++;
            time+=p.dt;
            Estart[0] = step;
            E1 = feltor.energy(), mass = feltor.mass(), diss = feltor.energy_diffusion();
            dEdt = (E1 - E0)/p.dt; 
            E0 = E1;
            accuracy = 2.*fabs( (dEdt-diss)/(dEdt + diss));
            evec = feltor.energy_vector();

            err = nc_open(argv[2], NC_WRITE, &ncid);
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &E1);
            err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass);
            for( unsigned i=0; i<3; i++)
            {

                err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);
            }
            err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
            err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
           
            err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt <<" Lambda = " << diss << " -> Accuracy: "<< accuracy << "\n";
            err = nc_close(ncid);

        }
#ifdef DG_BENCHMARK
        ti.toc();
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
        //////////////////////////write fields////////////////////////
        start[0] = i;
        err = nc_open(argv[2], NC_WRITE, &ncid);
        for( unsigned j=0; j<4; j++)
        {
            dg::blas2::symv( interpolate, y0[j], transferD);
            dg::blas1::transfer( transferD, transferH);
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = feltor.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[4], start, count, transferH.data() );
        transfer = feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);            
        dg::blas2::symv( interpolate,y1[1], transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[5], start, count, transferH.data() );

        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        err = nc_close(ncid);
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

