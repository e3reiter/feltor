#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
#include <csignal>
// #define DG_DEBUG

#include <mpi.h> //activate mpi

#include "netcdf_par.h"

#include "dg/algorithm.h"
#include "dg/backend/timer.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/interpolation.cuh"
#include "file/nc_utilities.h"

#include "feltor.cuh"
#include "parameters.h"


/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/


namespace ns_ncid
{
    int ncid;
}


void sigterm_handler(int signal)
{
    file :: NC_Error_Handle err;
    std::cout << "sigterm_handler, got signal " << signal << std::endl;
    std::cout << "ncid = " << ns_ncid :: ncid << std::endl;
    if(ns_ncid :: ncid != -1)
    {
        err = nc_close(ns_ncid :: ncid); 
        std::cerr << "SIGTERM caught. Closing NetCDF file with id " << ns_ncid :: ncid << std::endl;
    }
    MPI_Finalize();
    exit(signal);
}


int main( int argc, char* argv[])
{
     ////////////////////////////////setup MPI///////////////////////////////
    int provided;
    MPI_Init_thread( &argc, &argv, MPI_THREAD_FUNNELED, &provided);
    if( provided != MPI_THREAD_FUNNELED)
    {
        std::cerr << "wrong mpi-thread environment provided!\n";
        return -1;
    }
    int rank, size;
    MPI_Comm_rank( MPI_COMM_WORLD, &rank);
    MPI_Comm_size( MPI_COMM_WORLD, &size);
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Reader reader;
    Json::Value js;
    if( argc != 3 && argc != 4)
    {
        if(rank==0)std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n"; 
        if(rank==0)std::cerr << "Usage: "<<argv[0]<<" [input.txt] [output.nc] [input.nc] \n";
        return -1;
    }
    else 
    {
        std::ifstream is(argv[1]);
        reader.parse( is, js, false);
    }
    std::string input = js.toStyledString(); 
    const eule::Parameters p( js);
    if(rank==0)p.display( std::cout);
     ////////////////////////////////setup MPI///////////////////////////////
    int periods[2] = {false, false}; //non-, non-, periodic
    if( p.bc_x == dg::PER) periods[0] = true;
    if( p.bc_y == dg::PER) periods[1] = true;
    int np[2];
    if(rank==0)
    {
        std::cin>> np[0] >> np[1] ;
        std::cout << "Computing with "<<np[0]<<" x "<<np[1] << " = "<<size<<std::endl;
        assert( size == np[0]*np[1]);
    }
    MPI_Bcast( np, 2, MPI_INT, 0, MPI_COMM_WORLD);
    MPI_Comm comm;
    MPI_Cart_create( MPI_COMM_WORLD, 2, np, periods, true, &comm);
    ////////////////////////////// Install signal handler ///////////////////
    std::signal(SIGINT, sigterm_handler);
    std::signal(SIGTERM, sigterm_handler);
    //////////////////////////////////////////////////////////////
      //Make grid
    dg::MPIGrid2d grid(     0., p.lx, 0.,p.ly, p.n,     p.Nx,     p.Ny,     p.bc_x, p.bc_y, comm);
    dg::MPIGrid2d grid_out( 0., p.lx, 0.,p.ly, p.n_out, p.Nx_out, p.Ny_out, p.bc_x, p.bc_y, comm);  
    //create RHS 
    if(rank==0) std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::CartesianMPIGrid2d, dg::MDMatrix, dg::MDVec > feltor( grid, p); //initialize before rolkar!
    if(rank==0) std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::CartesianMPIGrid2d, dg::MDMatrix, dg::MDVec > rolkar( grid, p);
    if(rank==0) std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////
    dg::Gaussian init0( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
    dg::ExpProfX prof(p.nprofileamp, p.bgprofamp,p.invkappa);

    std::vector<dg::MDVec> y0(2, dg::evaluate( prof, grid)), y1(y0); 
    y1[1] = dg::evaluate( init0, grid);

    double time = 0;  
    if (argc ==3){
        if (p.modelmode==0 || p.modelmode==1)
        {
            dg::blas1::pointwiseDot(y1[1], y0[1],y1[1]); //<n>*ntilde
            dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni = <n> + <n>*ntilde
            dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //initialize ni-1
        }
        if (p.modelmode==2)
        {
            y0[1] = dg::evaluate( init0, grid);
        }
        if(rank==0) std::cout << "intiialize ne" << std::endl;
        feltor.initializene( y0[1], y0[0]);    
        if(rank==0) std::cout << "Done!\n";
    }
 if (argc==4) {
      file::NC_Error_Handle errIN;
      int ncidIN;
      errIN = nc_open( argv[3], NC_NOWRITE, &ncidIN);
      ///////////////////read in and show inputfile und geomfile//////////////////
      size_t lengthIN;
      errIN = nc_inq_attlen( ncidIN, NC_GLOBAL, "inputfile", &lengthIN);
      std::string inputIN( lengthIN, 'x');
      errIN = nc_get_att_text( ncidIN, NC_GLOBAL, "inputfile", &inputIN[0]);    
      std::cout << "input "<<inputIN<<std::endl;    
      const eule::Parameters pIN( js);
      pIN.display( std::cout);
      dg::MPIGrid2d grid_IN( 0., pIN.lx, 0., pIN.ly, pIN.n_out, pIN.Nx_out, pIN.Ny_out, pIN.bc_x, pIN.bc_y,comm);  
      int dimsIN[2],  coordsIN[2];
      MPI_Cart_get( comm, 2, dimsIN, periods, coordsIN);
      size_t count2dIN[3] = {1, grid_IN.n()*grid_IN.Ny(), grid_IN.n()*grid_IN.Nx()};  
      size_t start2dIN[3] = {0, coordsIN[1]*count2dIN[1], coordsIN[0]*count2dIN[2]}; 
      dg::HVec transferIN( dg::evaluate(dg::zero, grid_IN.local()));
      dg::DVec transferIND( dg::evaluate(dg::zero, grid_IN.local()));
      dg::IDMatrix interpolateIN = dg::create::interpolation( grid.local(),grid_IN.local()); 
      std::string namesIN[2] = {"electrons", "ions"};       
      int dataIDsIN[2];     
      int timeIDIN;
      double  timeIN;
      size_t stepsIN;
      /////////////////////The initial field///////////////////////////////////////////
      /////////////////////Get time length and initial data///////////////////////////
      errIN = nc_inq_varid(ncidIN, namesIN[0].data(), &dataIDsIN[0]);
      errIN = nc_inq_dimlen(ncidIN, dataIDsIN[0], &stepsIN);
      stepsIN-=1;
      start2dIN[0] = stepsIN/pIN.itstp;
      errIN = nc_inq_varid(ncidIN, "time", &timeIDIN);
      errIN = nc_get_vara_double( ncidIN, timeIDIN,start2dIN, count2dIN, &timeIN);
      if(rank==0) std::cout << "timein= "<< timeIN <<  std::endl;
      time=timeIN;
      errIN = nc_get_vara_double( ncidIN, dataIDsIN[0], start2dIN, count2dIN,transferIN.data());
       dg::blas1::transfer(transferIN,transferIND);
      dg::blas2::gemv( interpolateIN, transferIND,y0[0].data());
      errIN = nc_inq_varid(ncidIN, namesIN[1].data(), &dataIDsIN[1]);
      errIN = nc_get_vara_double( ncidIN, dataIDsIN[1], start2dIN, count2dIN, transferIN.data());
      dg::blas1::transfer(transferIN,transferIND);
      dg::blas2::gemv( interpolateIN, transferIND,y0[1].data());
      errIN = nc_close(ncidIN);
    }
   
    dg::Karniadakis< std::vector<dg::MDVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    if(rank==0) std::cout << "intialize Timestepper" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    if(rank==0) std::cout << "Done!\n";
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    MPI_Info info = MPI_INFO_NULL;
//         err = nc_create( argv[2],NC_NETCDF4|NC_CLOBBER, &ncid);//MPI OFF
    err = nc_create_par( argv[2], NC_NETCDF4|NC_MPIIO|NC_CLOBBER, comm, info, &ncid); //MPI ON
    ns_ncid :: ncid = ncid;
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    int dim_ids[3], tvarID;
    dg::Grid2d global_grid_out ( 0., p.lx, 0.,p.ly, p.n_out, p.Nx_out, p.Ny_out, p.bc_x, p.bc_y);  
    err = file::define_dimensions( ncid, dim_ids, &tvarID, global_grid_out);
    err = nc_enddef( ncid);
    err = nc_redef(ncid);

    //field IDs
    std::string names[4] = {"electrons", "ions",  "potential","vor"}; 
    int dataIDs[4]; 
    for( unsigned i=0; i<4; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 3, dim_ids, &dataIDs[i]);
        err = nc_var_par_access( ncid, dataIDs[i], NC_COLLECTIVE);
    }
    err = nc_var_par_access( ncid, tvarID, NC_COLLECTIVE);

    //energy IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    err = nc_var_par_access( ncid, EtimevarID, NC_COLLECTIVE);

    int energyID, massID, energyIDs[3], dissID, dEdtID, accuracyID;
    err = nc_def_var( ncid, "energy",   NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_var_par_access( ncid, energyID, NC_COLLECTIVE);
    err = nc_def_var( ncid, "mass",   NC_DOUBLE, 1, &EtimeID, &massID);
    err = nc_var_par_access( ncid, massID, NC_COLLECTIVE);

    std::string energies[3] = {"Se", "Si", "Uperp"}; 
    for( unsigned i=0; i<3; i++){
        err = nc_def_var( ncid, energies[i].data(), NC_DOUBLE, 1, &EtimeID, &energyIDs[i]);
        err = nc_var_par_access( ncid, energyIDs[i], NC_COLLECTIVE);
    }
    err = nc_def_var( ncid, "dissipation",   NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_var_par_access( ncid, dissID, NC_COLLECTIVE);
    err = nc_def_var( ncid, "dEdt",     NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_var_par_access( ncid, dEdtID, NC_COLLECTIVE);
    err = nc_def_var( ncid, "accuracy", NC_DOUBLE, 1, &EtimeID, &accuracyID);
    err = nc_var_par_access( ncid, accuracyID, NC_COLLECTIVE);
    //probe vars definition
    int NepID,phipID,radtransID,couplingID;
    err = nc_def_var( ncid, "Ne_p",     NC_DOUBLE, 1, &EtimeID, &NepID);
    err = nc_var_par_access( ncid, NepID, NC_COLLECTIVE);
    err = nc_def_var( ncid, "phi_p",    NC_DOUBLE, 1, &EtimeID, &phipID);  
    err = nc_var_par_access( ncid, phipID, NC_COLLECTIVE);
    err = nc_def_var( ncid, "G_nex",    NC_DOUBLE, 1, &EtimeID, &radtransID);
    err = nc_var_par_access( ncid, radtransID, NC_COLLECTIVE);
    err = nc_def_var( ncid, "Coupling",    NC_DOUBLE, 1, &EtimeID, &couplingID);  
    err = nc_var_par_access( ncid, couplingID, NC_COLLECTIVE);
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    if(rank==0) std::cout << "First output ... \n";
    int dims[2],  coords[2];
    MPI_Cart_get( comm, 2, dims, periods, coords);
    size_t count[3] = {1, grid_out.n()*grid_out.Ny(), grid_out.n()*grid_out.Nx()};  
    size_t start[3] = {0, coords[1]*count[1],          coords[0]*count[2]}; 
    dg::MDVec transfer( dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out.local()));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out.local()));
    dg::IDMatrix interpolate = dg::create::interpolation( grid_out.local(), grid.local()); //create local interpolation matrix
    for( unsigned i=0; i<2; i++)
    {
        dg::blas2::gemv( interpolate, y0[i].data(), transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    //pot
    transfer = feltor.potential()[0];
    dg::blas2::gemv( interpolate, transfer.data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
    //Vor
    transfer = feltor.potential()[0];
    dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);        
    dg::blas2::gemv( interpolate,y1[1].data(), transferD);
    dg::blas1::transfer( transferD, transferH);
    err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );

    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);

    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    double energy0 = feltor.energy(), mass0 = feltor.mass(), E0 = energy0, mass = mass0, E1 = 0.0, dEdt = 0., diss = 0., accuracy=0.;
    double Nep=0.;
    double phip=0.;
    double radtrans = feltor.radial_transport();
    double coupling = feltor.coupling();
    std::vector<double> evec = feltor.energy_vector();
    err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &energy0);
    err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass0);
    for( unsigned i=0; i<3; i++)
        err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);

    err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
    err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
    //probe
    err = nc_put_vara_double( ncid, NepID,      Estart, Ecount,&Nep);
    err = nc_put_vara_double( ncid, phipID,     Estart, Ecount,&phip);
    err = nc_put_vara_double( ncid, radtransID, Estart, Ecount,&radtrans);
    err = nc_put_vara_double( ncid, couplingID, Estart, Ecount,&coupling);
    err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);    
//     err = nc_close(ncid);
    if(rank==0) std::cout << "First write successful!\n";

    ///////////////////////////////////////Timeloop/////////////////////////////////

#ifdef DG_BENCHMARK
    dg::Timer t;
    t.tic();
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                if(rank==0)std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                if(rank==0)std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                MPI_Finalize();
                return -1;
            }
            step++;
            time+=p.dt;
            Estart[0] = step;
            E1 = feltor.energy(), mass = feltor.mass(), diss = feltor.energy_diffusion();
            dEdt = (E1 - E0)/p.dt; 
            E0 = E1;
            accuracy = 2.*fabs( (dEdt-diss)/(dEdt + diss));
            evec = feltor.energy_vector();
            radtrans = feltor.radial_transport();
            coupling= feltor.coupling();
            //err = nc_open(argv[2], NC_WRITE, &ncid);
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &E1);
            err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass);
            for( unsigned i=0; i<3; i++)
            {
                err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);
            }
            err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
            err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
            err = nc_put_vara_double( ncid, NepID,      Estart, Ecount,&Nep);
            err = nc_put_vara_double( ncid, phipID,     Estart, Ecount,&phip);         
            err = nc_put_vara_double( ncid, radtransID, Estart, Ecount,&radtrans);
            err = nc_put_vara_double( ncid, couplingID, Estart, Ecount,&coupling);    
            err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
            if(rank==0) std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass0<<"\t";
            if(rank==0) std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            if(rank==0) std::cout <<" d E/dt = " << dEdt <<" Lambda = " << diss << " -> Accuracy: "<< accuracy << "\n";
//             err = nc_close(ncid);

        }
#ifdef DG_BENCHMARK
        ti.toc();
        if(rank==0) std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        if(rank==0) std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
        ti.tic();
#endif//DG_BENCHMARK
        //////////////////////////write fields////////////////////////
        start[0] = i;
//         err = nc_open(argv[2], NC_WRITE, &ncid);
        for( unsigned j=0; j<2; j++)
        {
            dg::blas2::gemv( interpolate, y0[j].data(), transferD);
            dg::blas1::transfer( transferD, transferH);
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data() );
        }
        transfer = feltor.potential()[0];
        dg::blas2::gemv( interpolate, transfer.data(), transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
        transfer = feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);        
        dg::blas2::gemv( interpolate,y1[1].data(), transferD);
        dg::blas1::transfer( transferD, transferH);
        err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
//         err = nc_close(ncid);
#ifdef DG_BENCHMARK
        ti.toc();
        if(rank==0)std::cout << "\n\t Time for output: "<<ti.diff()<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
    }
#ifdef DG_BENCHMARK
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    if(rank==0) std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    if(rank==0) std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    if(rank==0) std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";
#endif//DG_BENCHMARK
    err = nc_close(ncid);
    ns_ncid :: ncid = -1;
    MPI_Finalize();

    return 0;

}

