#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG

#include "draw/host_window.h"
//#include "draw/device_window.cuh"
#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"

#include "feltor.cuh"
#include "parameters.h"
#include "../diag/probes.h"



/*
   - reads parameters from input.txt or any other given file, 
   - integrates the Feltor - functor and 
   - directly visualizes results on the screen using parameters in window_params.txt
*/


int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    Json::Reader reader;
    Json::Value js;
    if( argc == 1)
    {
        std::ifstream is("input.json");
        reader.parse(is,js,false);
    }
    else if( argc == 2)
    {
        std::ifstream is(argv[1]);
        reader.parse(is,js,false);
    }
    else
    {
        std::cerr << "ERROR: Too many arguments!\nUsage: "<< argv[0]<<" [filename]\n";
        return -1;
    }
    const eule::Parameters p(  js);    
    p.display( std::cout);
    /////////glfw initialisation ////////////////////////////////////////////
    std::stringstream title;
    std::ifstream is( "window_params.js");
    reader.parse( is, js, false);
    is.close();
    GLFWwindow* w = draw::glfwInitAndCreateWindow( js["cols"].asUInt()*js["width"].asUInt()*p.lx/p.ly, js["rows"].asUInt()*js["height"].asUInt(), "");
    draw::RenderHostData render(js["rows"].asUInt(), js["cols"].asUInt());
    //////////////////////////////////////////////////////////////////////////

    //Make grid
     dg::Grid2d grid( 0., p.lx, 0.,p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);  
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::CartesianGrid2d, dg::DMatrix, dg::DVec > feltor( grid, p); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::CartesianGrid2d, dg::DMatrix, dg::DVec > rolkar( grid, p);
    std::cout << "Done!\n";

    /////////////////////The initial field///////////////////////////////////////////

    dg::ExpProfX prof(p.nprofileamp, p.bgprofamp,p.invkappa);
    std::vector<dg::DVec> y0(2, dg::evaluate( prof, grid)), y1(y0); 

    
    if (p.initmode == 0) { 
      dg::Gaussian init0( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp);
      y1[1] = dg::evaluate( init0, grid);
    }
    if (p.initmode == 1) {
      dg::SinXSinY init0(p.amp,0.,2*M_PI/p.lx,p.sigma*2*M_PI/p.ly);
      y1[1] = dg::evaluate( init0, grid);
    }
    if (p.initmode == 2) {
      dg::BathRZ init0(16,16,1.,0.,0., 30.,5.,p.amp);
      y1[1] = dg::evaluate( init0, grid);
      dg::DVec  dampr = dg::evaluate(dg::TanhProfX(p.lx*0.95,p.sourcew,-1.0,0.0,1.0),grid);
      dg::DVec  dampl = dg::evaluate(dg::TanhProfX(p.lx*0.05,p.sourcew,1.0,0.0,1.0),grid);
      dg::blas1::pointwiseDot(y1[1],dampr,y1[1]);
      dg::blas1::pointwiseDot(y1[1],dampl,y1[1]);
   
    }  
        
    if (p.modelmode == 0 || p.modelmode == 1)
    {
        dg::blas1::pointwiseDot(y1[1], y0[1],y1[1]); //<n>*ntilde
        dg::blas1::axpby( 1., y1[1], 1., y0[1]); //initialize ni = <n> + <n>*ntilde
        dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //initialize ni-1
        std::cout << "intiialize ne" << std::endl;
        feltor.initializene( y0[1], y0[0]);    
        std::cout << "Done!\n";
    }
    if (p.modelmode == 2) {
        std::cout << "intiialize ne" << std::endl;
        dg::blas1::axpby(1.0,y1[1],0.,y0[1],y0[1]);
        feltor.initializene( y1[1], y0[0]);    
        std::cout << "Done!\n";
    }
    if (p.modelmode == 3) {
        dg::blas1::pointwiseDot( y0[1],y1[1],y0[1]); //<n>*Ntilde
        dg::blas1::axpby( 1., y0[1], 1.,y1[0], y0[1]); //initialize Ni = <n> + <n>*Ntilde
        dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //initialize Ni-1
        
        std::cout << "intiialize ne" << std::endl;
        feltor.initializene( y0[1], y0[0]); //n_e-1
        
        
        dg::blas1::transform( y1[1], y0[1], dg::PLUS<>(+1.0)); // (1+Nitilde)
        dg::blas1::transform( y0[1], y0[1], dg::LN<double>()); //ln (1+Nitilde)
        
        dg::blas1::transform(y0[0], y0[0], dg::PLUS<>((p.bgprofamp + p.nprofileamp))); //ne
        dg::blas1::pointwiseDivide(y0[0], y1[0],y0[0]); // 1+ netilde
        dg::blas1::transform( y0[0], y0[0], dg::LN<double>()); //ln (1+netilde)

        std::cout << "Done!\n";
    } 



    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    std::cout << "intiialize karniadakis" << std::endl;
    karniadakis.init( feltor, rolkar, y0, p.dt);
    std::cout << "Done!\n";

    dg::DVec dvisual( grid.size(), 0.);
    dg::DVec dvisual2( grid.size(), 0.);
    dg::HVec hvisual( grid.size(), 0.), visual(hvisual),avisual(hvisual);
    dg::IHMatrix equi = dg::create::backscatter( grid);
    draw::ColorMapRedBlueExtMinMax colors(-1.0, 1.0);
    //create timer
    dg::Timer t;
    double time = 0;
    unsigned step = 0;
    
    const double mass0 = feltor.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = feltor.energy(), energy0 = E0, E1 = 0., diff = 0.;
    
    std::cout << "Begin computation \n";
    std::cout << std::scientific << std::setprecision( 2);
    
    dg::DVec xprobecoords(7,1.);
    for (unsigned i=0;i<7; i++) {
        xprobecoords[i] = p.lx/8.*(1+i) ;
    }
    const dg::DVec yprobecoords(7,p.ly/2.);
    probes<dg::IDMatrix,dg::DMatrix, dg::DVec> pro(xprobecoords,yprobecoords,grid);
    while ( !glfwWindowShouldClose( w ))
    {

        dg::blas1::transfer(y0[0], hvisual);
//         if 
//         dg::blas1::axpby(1.0,hvisual,
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(), (double)-1e14, thrust::maximum<double>() );
//         colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() = 1.0;
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ne / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ne-1 / " << colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        //draw ions
        //thrust::transform( y1[1].begin(), y1[1].end(), dvisual.begin(), dg::PLUS<double>(-0.));//ne-1
        dg::blas1::transfer(y0[1], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
//         colors.scalemin() = -colors.scalemax();        
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );

        title << std::setprecision(2) << std::scientific;
        //title <<"ni / "<<(double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() )<<"  " << colors.scalemax()<<"\t";
        title <<"ni-1 / " << colors.scalemin()<<"\t";

        render.renderQuad(visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);

        
        //draw potential
        //transform to Vor
//        dvisual=feltor.potential()[0];
//        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
//        hvisual = y1[1];
        dg::blas1::transfer(feltor.potential()[0], hvisual);
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );

        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax() ,thrust::minimum<double>() );

//         //colors.scalemin() = 1.0;        
//          colors.scalemin() = -colors.scalemax();        
//          colors.scalemin() = -colors.scalemax();        
        //colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Potential / "<< colors.scalemax() << " " << colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);
        //draw potential
        //transform to Vor
        dvisual=feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), dvisual, y1[1]);
        dg::blas1::transfer(y1[1], hvisual);
         //hvisual = feltor.potential()[0];
        dg::blas2::gemv( equi, hvisual, visual);
        colors.scalemax() = (double)thrust::reduce( visual.begin(), visual.end(),  (double)-1e14, thrust::maximum<double>() );
        //colors.scalemin() = 1.0;        
//          colors.scalemin() = -colors.scalemax();        
        colors.scalemin() =  (double)thrust::reduce( visual.begin(), visual.end(), colors.scalemax()  ,thrust::minimum<double>() );
        title <<"Omega / "<< colors.scalemax()<< " "<< colors.scalemin()<<"\t";

        render.renderQuad( visual, grid.n()*grid.Nx(), grid.n()*grid.Ny(), colors);


     
           
        title << std::fixed; 
        title << " &&   time = "<<time;
        glfwSetWindowTitle(w,title.str().c_str());
        title.str("");
        glfwPollEvents();
        glfwSwapBuffers( w);

        //step 
#ifdef DG_BENCHMARK
        t.tic();
#endif//DG_BENCHMARK
        for( unsigned i=0; i<p.itstp; i++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                glfwSetWindowShouldClose( w, GL_TRUE);
                break;
            }
            step++;
            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass_blob0<<"\t";
            E1 = feltor.energy();
            diff = (E1 - E0)/p.dt; //
            double diss = feltor.energy_diffusion( );
//             double coupling = feltor.coupling();
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout << 
                         " Charge= " << feltor.charge() <<
                         " Accuracy: "<< 2.*fabs((diff-diss)/(diff+diss))<<
                         " d E/dt = " << diff <<
                         " Lambda =" << diss <<  std::endl;
            E0 = E1;
        }
        dg::blas1::transform( y0[0], dvisual, dg::PLUS<>(+(p.bgprofamp + p.nprofileamp))); //npe = N+1
        dvisual2 = feltor.potential()[0];
        pro.fluxes(time,  dvisual,dvisual2);
        pro.profiles(time,dvisual,dvisual2);
//         p.profiles
        time += (double)p.itstp*p.dt;
#ifdef DG_BENCHMARK
        t.toc();
        std::cout << "\n\t Step "<<step;
        std::cout << "\n\t Average time for one step: "<<t.diff()/(double)p.itstp<<"s\n\n";
#endif//DG_BENCHMARK
    }
    glfwTerminate();
    ////////////////////////////////////////////////////////////////////

    return 0;

}
