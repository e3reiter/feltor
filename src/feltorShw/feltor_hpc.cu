#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <sstream>
#include <cmath>
// #define DG_DEBUG



#include "dg/backend/xspacelib.cuh"
#include "dg/backend/timer.cuh"

#include "dg/backend/interpolation.cuh"
#include "file/read_input.h"
#include "file/nc_utilities.h"

#include "feltor.cuh"
#include "parameters.h"


/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

int main( int argc, char* argv[])
{
    ////////////////////////Parameter initialisation//////////////////////////
    std::vector<double> v,v3;
    std::string input;
    if( argc != 3 && argc != 4)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n"; 
        std::cerr << "Usage: "<<argv[0]<<" [input.txt] [output.nc] [input.nc] \n";
        return -1;
    }
    else 
    {
        input = file::read_file( argv[1]); //deprecated, better use json reader directly, instead!
    }
    Json::Reader reader;
    Json::Value js;
    reader.parse( input, js, false);
    std::cout << js<<std::endl;
    input = js.toStyledString(); //save input without comments, which is important if netcdf file is later read by another parser
    const eule::Parameters p( js);
    p.display( std::cout);

    //Make grid
    dg::Grid2d grid( 0., p.lx, 0., p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    dg::Grid2d grid_out( 0., p.lx, 0., p.ly, p.n_out, p.Nx_out, p.Ny_out, p.bc_x, p.bc_y);  
    // Grid for radial probe location. This is used only in netcdf output, probe positioning is still hard-coded
    dg::Grid1d grid_probe(0, p.lx, 1, 8, p.bc_x);
    //create RHS 
    std::cout << "Constructing Feltor...\n";
    eule::Feltor<dg::CartesianGrid2d, dg::DMatrix, dg::DVec > feltor( grid, p); //initialize before rolkar!
    std::cout << "Constructing Rolkar...\n";
    eule::Rolkar<dg::CartesianGrid2d, dg::DMatrix, dg::DVec > rolkar( grid, p);
    std::cout << "Done!\n";
    /////////////////////The initial field///////////////////////////////////////////
    //initial perturbation
    //dg::Gaussian3d init0(gp.R_0+p.posX*gp.a, p.posY*gp.a, M_PI, p.sigma, p.sigma, p.sigma, p.amp);
    dg::Gaussian init0(p.posX * p.lx, p.posY * p.ly, p.sigma, p.sigma, p.amp);
//     dg::BathRZ init0(16, 16, 1, 2.0, 2.0, 30.0, 5.0, p.amp);
//     solovev::ZonalFlow init0(p, gp);
//     dg::CONSTANT init0( 0.);    
    //background profile
//     solovev::Nprofile prof(p, gp); //initial background profile
//     dg::CONSTANT prof(p.bgprofamp );
    //
//     dg::LinearX prof(-p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);
//     dg::SinProfX prof(p.nprofileamp, p.bgprofamp,M_PI/(2.*p.lx));
    dg::ExpProfX prof(p.nprofileamp, p.bgprofamp,p.ln);
//     dg::TanhProfX prof(p.lx*p.solb,p.ln,-1.0,p.bgprofamp,p.nprofileamp); //<n>
//     dg::TanhProfX prof(p.lx*p.solb,p.lx/10.,-1.0,p.bgprofamp,p.nprofileamp); //<n>

//     const dg::DVec prof =  dg::LinearX( -p.nprofileamp/((double)p.lx), p.bgprofamp + p.nprofileamp);

    std::vector<dg::DVec> y0(2, dg::evaluate(prof, grid)), y1(y0); 
    dg::HVec temp(dg::evaluate(dg::zero,grid));
    double time = 0;

    if (argc ==3){
      y1[1] = dg::evaluate( init0, grid);
      dg::blas1::pointwiseDot(y1[1], y0[1], y1[1]);

      dg::blas1::axpby(1., y1[1], 1., y0[1]); //initialize ni
      dg::blas1::transform(y0[1], y0[1], dg::PLUS<>(-(p.bgprofamp + p.nprofileamp))); //initialize ni-1
      std::cout << "intiialize ne" << std::endl;
      feltor.initializene(y0[1], y0[0]);    
      std::cout << "Done!\n";
    }
    if (argc==4) {
      file::NC_Error_Handle errIN;
      int ncidIN;
      errIN = nc_open( argv[3], NC_NOWRITE, &ncidIN);
      ///////////////////read in and show inputfile und geomfile//////////////////
      size_t lengthIN;
      errIN = nc_inq_attlen( ncidIN, NC_GLOBAL, "inputfile", &lengthIN);
      std::string inputIN( lengthIN, 'x');
      errIN = nc_get_att_text( ncidIN, NC_GLOBAL, "inputfile", &inputIN[0]);    
      std::cout << "input "<<inputIN<<std::endl;    
      const eule::Parameters pIN(  js);    
      pIN.display( std::cout);
      dg::Grid2d grid_IN( 0., pIN.lx, 0., pIN.ly, pIN.n_out, pIN.Nx_out, pIN.Ny_out, pIN.bc_x, pIN.bc_y);  
      dg::HVec transferINH( dg::evaluate(dg::zero, grid_IN));
      size_t count2dIN[3]  = {1, grid_IN.n()*grid_IN.Ny(), grid_IN.n()*grid_IN.Nx()};
      size_t start2dIN[3]  = {0, 0, 0};
      std::string namesIN[2] = {"electrons", "ions"}; 
      
      int dataIDsIN[2];     
      int timeIDIN;
      double  timeIN;
      size_t stepsIN;
      /////////////////////The initial field///////////////////////////////////////////
      /////////////////////Get time length and initial data///////////////////////////
      errIN = nc_inq_varid(ncidIN, namesIN[0].data(), &dataIDsIN[0]);
      errIN = nc_inq_dimlen(ncidIN, dataIDsIN[0], &stepsIN);
      stepsIN-=1;
      start2dIN[0] = stepsIN/pIN.itstp;
      std::cout << "stepsin= "<< stepsIN <<  std::endl;
      std::cout << "start2dIN[0]= "<< start2dIN[0] <<  std::endl;
      errIN = nc_inq_varid(ncidIN, "time", &timeIDIN);
      errIN = nc_get_vara_double( ncidIN, timeIDIN,start2dIN, count2dIN, &timeIN);
      std::cout << "timein= "<< timeIN <<  std::endl;
      time=timeIN;
      dg::IHMatrix interpolateIN = dg::create::interpolation( grid,grid_IN); 
      errIN = nc_get_vara_double( ncidIN, dataIDsIN[0], start2dIN, count2dIN, transferINH.data());
      dg::blas2::gemv( interpolateIN, transferINH,temp);
      dg::blas1::transfer(temp,y0[0]);
      errIN = nc_inq_varid(ncidIN, namesIN[1].data(), &dataIDsIN[1]);
      errIN = nc_get_vara_double( ncidIN, dataIDsIN[1], start2dIN, count2dIN, transferINH.data());
      dg::blas2::gemv( interpolateIN, transferINH,temp);
      dg::blas1::transfer(temp,y0[1]);      
      errIN = nc_close(ncidIN);

    }

    dg::Karniadakis< std::vector<dg::DVec> > karniadakis( y0, y0[0].size(), p.eps_time);
    karniadakis.init( feltor, rolkar, y0, p.dt);
//     feltor.energies( y0);//now energies and potential are at time 0
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[2], NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    int dim_ids_field[3], tvarID_field;
    err = file::define_dimensions(ncid, dim_ids_field, &tvarID_field, grid_out);
    err = nc_enddef(ncid);
    err = nc_redef(ncid);

    //field IDs
    std::vector<std::string> varname_fields;
    varname_fields.push_back("electrons"); varname_fields.push_back("ions"); varname_fields.push_back("potential"); varname_fields.push_back("vor");
    int dataIDs[4]; 
    for(unsigned i = 0; i < varname_fields.size(); i++)
        err = nc_def_var(ncid, varname_fields[i].data(), NC_DOUBLE, 3, dim_ids_field, &dataIDs[i]);
    //energy IDs, used for small time-step diagnostic
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, energyIDs[3], dissID, dEdtID, accuracyID, couplingID,radtransID; 

    err = nc_def_var(ncid, "energy", NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var(ncid, "mass", NC_DOUBLE, 1, &EtimeID, &massID);
    err = nc_def_var(ncid, "dissipation", NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var(ncid, "dEdt", NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_def_var(ncid, "accuracy", NC_DOUBLE, 1, &EtimeID, &accuracyID);
    err = nc_def_var(ncid, "Coupling", NC_DOUBLE, 1, &EtimeID, &couplingID);  
    err = nc_def_var(ncid, "radtrans", NC_DOUBLE, 1, &EtimeID, &radtransID);  

    std::string energies[3] = {"Se", "Si", "Uperp"}; 
    for(unsigned i = 0; i < 3; i++)
    {
        err = nc_def_var( ncid, energies[i].data(), NC_DOUBLE, 1, &EtimeID, &energyIDs[i]);
    }

   // Probe IDs
    std::vector<std::string> varname_probes;
    varname_probes.push_back("probe_ne"); varname_probes.push_back("probe_phi"); varname_probes.push_back("probe_Gamma_x");
    // Create x-dimension for probe 

    int ID_probes[3];
    int dim_ids_probe[2];
    dim_ids_probe[0] = EtimeID;
    //dim_ids_probe[1] = 
    file :: define_dimension(ncid, "X_probe", &dim_ids_probe[1], dg::evaluate(dg::LinearX(1.0, 0), grid_probe).data(), 8);
    for(unsigned i = 0; i < varname_probes.size(); i++)
    {
        err = nc_def_var(ncid, varname_probes[i].data(), NC_DOUBLE, 2, dim_ids_probe, &ID_probes[i]);
    } 
    
    err = nc_enddef(ncid);
    ///////////////////////////////////first output/////////////////////////
    std::cout << "First output ... \n";
    size_t start[3] = {0, 0, 0};
    size_t count[3] = {1, grid_out.n() * grid_out.Ny(), grid_out.n() * grid_out.Nx()};
    dg::DVec transfer( dg::evaluate(dg::zero, grid));
    dg::DVec transferD( dg::evaluate(dg::zero, grid_out));
    dg::HVec transferH( dg::evaluate(dg::zero, grid_out));
    dg::IDMatrix interpolate = dg::create::interpolation( grid_out, grid); 
    for( unsigned i=0; i<2; i++)
    {
        dg::blas2::gemv( interpolate, y0[i], transferD);
        dg::blas1::transfer( transferD, transferH); //transfer to host
        err = nc_put_vara_double( ncid, dataIDs[i], start, count, transferH.data() );
    }
    //pot
    transfer = feltor.potential()[0];
    dg::blas2::symv( interpolate, transfer, transferD);
    dg::blas1::transfer( transferD, transferH); //transfer to host

    err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
    //Vor
    transfer = feltor.potential()[0];
    dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);            
    dg::blas2::symv( interpolate,y1[1], transferD);
    dg::blas1::transfer( transferD, transferH); //transfer to host
    err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data() );



    err = nc_put_vara_double( ncid, tvarID_field, start, count, &time);
    err = nc_put_vara_double( ncid, EtimevarID, start, count, &time);

    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    double energy0 = feltor.energy(), mass0 = feltor.mass(), E0 = energy0, mass = mass0, E1 = 0.0, dEdt = 0., diss = 0., accuracy=0.;
    double radtrans = feltor.radial_transport();
    double coupling = feltor.coupling();
    std::vector<double> evec = feltor.energy_vector();
    err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &energy0);
    err = nc_put_vara_double( ncid, massID, Estart, Ecount, &mass0);
    for( unsigned i=0; i<3; i++)
        err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);

    err = nc_put_vara_double( ncid, dissID, Estart, Ecount, &diss);
    err = nc_put_vara_double( ncid, dEdtID, Estart, Ecount, &dEdt);

    err = nc_put_vara_double( ncid, couplingID, Estart, Ecount, &coupling);
    err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount, &accuracy);
    err = nc_put_vara_double( ncid, radtransID, Estart, Ecount, &radtrans);

    err = nc_close(ncid);
    std::cout << "First write successful!\n";

    ///////////////////////////////////////Timeloop/////////////////////////////////
    dg::Timer t;
    t.tic();
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            try{ karniadakis( feltor, rolkar, y0);}
            catch( dg::Fail& fail) { 
                std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
                std::cerr << "Does Simulation respect CFL condition?\n";
                err = nc_close(ncid);
                return -1;
            }
            step++;
            time+=p.dt;
            Estart[0] = step;
            E1 = feltor.energy(), mass = feltor.mass(), diss = feltor.energy_diffusion();
            dEdt = (E1 - E0)/p.dt; 
            E0 = E1;
            accuracy = 2.*fabs( (dEdt-diss)/(dEdt + diss));
            evec = feltor.energy_vector();
            radtrans = feltor.radial_transport();
            coupling= feltor.coupling();
            err = nc_open(argv[2], NC_WRITE, &ncid);
            err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
            err = nc_put_vara_double( ncid, energyID, Estart, Ecount, &E1);
            err = nc_put_vara_double( ncid, massID,   Estart, Ecount, &mass);
            for( unsigned i=0; i<3; i++)
            {
                err = nc_put_vara_double( ncid, energyIDs[i], Estart, Ecount, &evec[i]);
            }
            err = nc_put_vara_double( ncid, dissID,     Estart, Ecount,&diss);
            err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount,&dEdt);
            err = nc_put_vara_double( ncid, couplingID, Estart, Ecount,&coupling);    
            err = nc_put_vara_double( ncid, accuracyID, Estart, Ecount,&accuracy);
            err = nc_put_vara_double( ncid, radtransID, Estart, Ecount,&radtrans);


            std::cout << "(m_tot-m_0)/m_0: "<< (feltor.mass()-mass0)/mass0<<"\t";
            std::cout << "(E_tot-E_0)/E_0: "<< (E1-energy0)/energy0<<"\t";
            std::cout <<" d E/dt = " << dEdt <<" Lambda = " << diss << " -> Accuracy: "<< accuracy << "\n";
            err = nc_close(ncid);
        }
#ifdef DG_BENCHMARK
        ti.toc();
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
        //////////////////////////write fields////////////////////////
        start[0] = i;
        err = nc_open(argv[2], NC_WRITE, &ncid);
        for(unsigned j = 0; j < 2; j++)
        {
            dg::blas2::symv( interpolate, y0[j], transferD);
            dg::blas1::transfer( transferD, transferH); //transfer to host
            err = nc_put_vara_double( ncid, dataIDs[j], start, count, transferH.data());
        }
        transfer = feltor.potential()[0];
        dg::blas2::symv( interpolate, transfer, transferD);
        dg::blas1::transfer( transferD, transferH); //transfer to host
        err = nc_put_vara_double( ncid, dataIDs[2], start, count, transferH.data() );
        transfer = feltor.potential()[0];
        dg::blas2::gemv( rolkar.laplacianM(), transfer, y1[1]);            
        dg::blas2::symv( interpolate,y1[1], transferD);
        dg::blas1::transfer( transferD, transferH); //transfer to host
        err = nc_put_vara_double( ncid, dataIDs[3], start, count, transferH.data());

        err = nc_put_vara_double(ncid, tvarID_field, start, count, &time);
        err = nc_close(ncid);
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;
}

