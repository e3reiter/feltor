#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>


#include "file/file.h"
#include "file/read_input.h"
#include "file/nc_utilities.h"

#include "toeflR.cuh"
#include "dg/algorithm.h"
#include "dg/backend/xspacelib.cuh"
#include "parameters.h"

#include "dg/backend/timer.cuh"


/*
   - reads parameters from input.txt or any other given file, 
   - integrates the ToeflR - functor and 
   - writes outputs to a given outputfile using hdf5. 
        density fields are the real densities in XSPACE ( not logarithmic values)
*/

int main( int argc, char* argv[])
{
    //Parameter initialisation
    std::vector<double> v;
    Json::Reader reader;
    Json::Value js;
    if( argc != 3)
    {
        std::cerr << "ERROR: Wrong number of arguments!\nUsage: "<< argv[0]<<" [inputfile] [outputfile]\n";
        return -1;
    }
    else 
    {
        std::ifstream is(argv[1]);
        reader.parse( is, js, false); //read input without comments
    }
    std::cout << js<<std::endl;
    std::string input = js.toStyledString(); //save input without comments, which is important if netcdf file is later read by another parser
    const Parameters p( js);
    p.display( std::cout);

    ////////////////////////////////set up computations///////////////////////////
    dg::Grid2d grid( 0, p.lx, 0, p.ly, p.n, p.Nx, p.Ny, p.bc_x, p.bc_y);
    //create RHS 
    dg::ToeflR< dg::CartesianGrid2d, dg::DMatrix, dg::DVec > test( grid, p); 
    dg::Diffusion<dg::CartesianGrid2d, dg::DMatrix, dg::DVec> diffusion( grid, p.nu);
    /////////////////////create initial vector////////////////////////////////////
    dg::Gaussian g( p.posX*p.lx, p.posY*p.ly, p.sigma, p.sigma, p.amp); 
    std::vector<dg::DVec> y0(2, dg::evaluate( g, grid)), y1(y0); // n_e' = gaussian
    dg::blas2::symv( test.gamma(), y0[0], y0[1]); // n_e = \Gamma_i n_i -> n_i = ( 1+alphaDelta) n_e' + 1
    {
        dg::DVec v2d = dg::create::inv_weights(grid);
        dg::blas2::symv( v2d, y0[1], y0[1]);
    }
    if( p.equations == "gravity_local" || p.equations == "gravity_global" || p.equations == "drift_global"){
        y0[1] = dg::evaluate( dg::zero, grid);
    }

    //////////////////initialisation of timestepper and first step///////////////////
    std::cout << "init timestepper...\n";
    double time = 0;
    //dg::AB< k, std::vector<dg::DVec> > ab( y0);
    dg::Karniadakis< std::vector<dg::DVec> > ab( y0, y0[0].size(), 1e-9);
    ab.init( test, diffusion, y0, p.dt);
    y0.swap( y1); //y1 now contains value at zero time
    /////////////////////////////set up netcdf/////////////////////////////////////
    file::NC_Error_Handle err;
    int ncid;
    err = nc_create( argv[2],NC_NETCDF4|NC_CLOBBER, &ncid);
    err = nc_put_att_text( ncid, NC_GLOBAL, "inputfile", input.size(), input.data());
    int dim_ids[3], tvarID;
    err = file::define_dimensions( ncid, dim_ids, &tvarID, grid);
    //field IDs
    std::string names[4] = {"electrons", "ions", "potential", "vorticity"}; 
    int dataIDs[4]; 
    for( unsigned i=0; i<4; i++){
        err = nc_def_var( ncid, names[i].data(), NC_DOUBLE, 3, dim_ids, &dataIDs[i]);}

    //energy IDs
    int EtimeID, EtimevarID;
    err = file::define_time( ncid, "energy_time", &EtimeID, &EtimevarID);
    int energyID, massID, dissID, dEdtID;
    err = nc_def_var( ncid, "energy",      NC_DOUBLE, 1, &EtimeID, &energyID);
    err = nc_def_var( ncid, "mass",        NC_DOUBLE, 1, &EtimeID, &massID);
    err = nc_def_var( ncid, "dissipation", NC_DOUBLE, 1, &EtimeID, &dissID);
    err = nc_def_var( ncid, "dEdt",        NC_DOUBLE, 1, &EtimeID, &dEdtID);
    err = nc_enddef(ncid);
    size_t start[3] = {0, 0, 0};
    size_t count[3] = {1, grid.n()*grid.Ny(), grid.n()*grid.Nx()};
    size_t Estart[] = {0};
    size_t Ecount[] = {1};
    ///////////////////////////////////first output/////////////////////////
    //output all three fields
    std::vector<dg::DVec> transferD(4);
    std::vector<dg::HVec> output(4);
    transferD[0] = y1[0], transferD[1] = y1[1], transferD[2] = test.potential()[0], transferD[3] = test.potential()[0]; //electrons
    start[0] = 0;
    for( int k=0;k<4; k++)
    {
        dg::blas1::transfer( transferD[k], output[k]);
        err = nc_put_vara_double( ncid, dataIDs[k], start, count, output[k].data() );
    }
    err = nc_put_vara_double( ncid, tvarID, start, count, &time);
    err = nc_close(ncid);
    ///////////////////////////////////////Timeloop/////////////////////////////////
    const double mass0 = test.mass(), mass_blob0 = mass0 - grid.lx()*grid.ly();
    double E0 = test.energy(), E1 = 0, diff = 0;
    dg::Timer t;
    t.tic();
    try
    {
#ifdef DG_BENCHMARK
    unsigned step = 0;
#endif //DG_BENCHMARK
    for( unsigned i=1; i<=p.maxout; i++)
    {

#ifdef DG_BENCHMARK
        dg::Timer ti;
        ti.tic();
#endif//DG_BENCHMARK
        for( unsigned j=0; j<p.itstp; j++)
        {
            ab( test, diffusion, y0);
            y0.swap( y1); //attention on -O3 ?
            //store accuracy details
            {
                std::cout << "(m_tot-m_0)/m_0: "<< (test.mass()-mass0)/mass_blob0<<"\t";
                E0 = E1;
                E1 = test.energy();
                diff = (E1 - E0)/p.dt;
                double diss = test.energy_diffusion( );
                std::cout << "diff diss: "<< diff<<" "<<diss<<"\t";
                std::cout << "Accuracy: "<< 2.*(diff-diss)/(diff+diss)<<"\n";
            }
            time+=p.dt;
            Estart[0] += 1;
            {
                err = nc_open(argv[2], NC_WRITE, &ncid);
                double ener=test.energy(), mass=test.mass(), diff=test.mass_diffusion(), dEdt=test.energy_diffusion();
                err = nc_put_vara_double( ncid, EtimevarID, Estart, Ecount, &time);
                err = nc_put_vara_double( ncid, energyID,   Estart, Ecount, &ener);
                err = nc_put_vara_double( ncid, massID,     Estart, Ecount, &mass);
                err = nc_put_vara_double( ncid, dissID,     Estart, Ecount, &diff);
                err = nc_put_vara_double( ncid, dEdtID,     Estart, Ecount, &dEdt);
                err = nc_close(ncid);
            }
        }
        //output all three fields and vorticity
        transferD[0] = y1[0], transferD[1] = y1[1], transferD[2] = test.potential()[0]; //electrons
        dg::blas2::symv( diffusion.laplacianM(), transferD[2], transferD[3]);
        for( int k=0;k<4; k++)
            dg::blas1::transfer( transferD[k], output[k]);
        err = nc_open(argv[2], NC_WRITE, &ncid);
        start[0] = i;
        for( int k=0; k<4; k++)
            err = nc_put_vara_double( ncid, dataIDs[k], start, count, output[k].data() );
        err = nc_put_vara_double( ncid, tvarID, start, count, &time);
        err = nc_close(ncid);

#ifdef DG_BENCHMARK
        ti.toc();
        step+=p.itstp;
        std::cout << "\n\t Step "<<step <<" of "<<p.itstp*p.maxout <<" at time "<<time;
        std::cout << "\n\t Average time for one step: "<<ti.diff()/(double)p.itstp<<"s\n\n"<<std::flush;
#endif//DG_BENCHMARK
    }
    }
    catch( dg::Fail& fail) { 
        std::cerr << "CG failed to converge to "<<fail.epsilon()<<"\n";
        std::cerr << "Does Simulation respect CFL condition?\n";
    }
    t.toc(); 
    unsigned hour = (unsigned)floor(t.diff()/3600);
    unsigned minute = (unsigned)floor( (t.diff() - hour*3600)/60);
    double second = t.diff() - hour*3600 - minute*60;
    std::cout << std::fixed << std::setprecision(2) <<std::setfill('0');
    std::cout <<"Computation Time \t"<<hour<<":"<<std::setw(2)<<minute<<":"<<second<<"\n";
    std::cout <<"which is         \t"<<t.diff()/p.itstp/p.maxout<<"s/step\n";

    return 0;

}

